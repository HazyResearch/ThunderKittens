#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include <iomanip>

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;

using c_dtype = float;

struct matmul_layout {
    // tiles for the quantized inputs
    using  a_tile   = st_fl8_e4m3<64, 128>; 
    using  b_tile   = st_fl8_e4m3<128, 128>;
    using  c_tile   = st<c_dtype, 64, 128>;
    using  a_layout = gl<fp8e4m3, 1, 1, -1, -1, a_tile>;
    using  b_layout = gl<fp8e4m3, 1, 1, -1, -1, b_tile>;
    using  c_layout = gl<c_dtype, 1, 1, -1, -1, c_tile>;

    // tiles for the dequantized inputs
    using scale_a_layout = gl<c_dtype, 1, 1, 1, -1>;
    using scale_b_layout = gl<c_dtype, 1, 1, 1, -1>;

    template<typename T=float> using accum_tile = rt<T, 16, c_tile::cols>;

    struct globals        { 
        a_layout A; b_layout B; c_layout C; 
        scale_a_layout scale_a; scale_b_layout scale_b;
    };

    struct input_block    { 
        a_tile a[2]; b_tile b; 
    };
    struct finish_block   { 
        c_tile c[2]; 
    };
    struct scratch_block  {
    };
    struct common_state   { int2 coord; };
    struct consumer_state { 
        accum_tile<c_dtype> accum;      // Changed to single tall accumulator
    };
};

template<int _SUPER_M=12>
struct matmul_template {
    static constexpr int SUPER_M = _SUPER_M;
    using layout    = matmul_layout;
    static constexpr int NUM_CONSUMER_WARPS=8, INPUT_PIPE_STAGES=4, PRODUCER_BARRIER_ARRIVALS=1;
    // Helper functions
    template<bool PERISISTENT_GRID=true> __host__ static inline dim3 grid(int M, int N, int K) {
        return dim3(PERISISTENT_GRID ? 132 : M*N/(2*layout::c_tile::num_elements));
    }
    // ThunderKittens template functions
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int Rblocks = args.globals.C.rows() / (2*layout::c_tile::rows), Cblocks = args.globals.C.cols() / layout::c_tile::cols;
        int super_rows = (Rblocks/SUPER_M)*SUPER_M,
            final_rows = Rblocks - super_rows,
            super_repeat = SUPER_M*Cblocks;
        int task_id = args.task_iter*gridDim.x + blockIdx.x;
        if (task_id < super_rows * Cblocks)
            args.common.coord = { SUPER_M*(task_id/super_repeat) + task_id%SUPER_M, (task_id%super_repeat)/SUPER_M };
        else if (task_id < Rblocks*Cblocks) {
            int remainder_id = task_id - super_rows*Cblocks;
            args.common.coord = { super_rows + (remainder_id%final_rows), remainder_id/final_rows };
        }
        else { // Id is too high, no more work to do
            args.num_iters = -1;
            return;
        }
        args.num_iters = args.globals.A.cols()/layout::a_tile::cols;
        int id = warpgroup::groupid() == NUM_CONSUMER_WARPS/4 ? 0 : warpgroup::groupid();
        args.common.coord = { args.common.coord.x*2 + id, args.common.coord.y };
    }

    struct producer {
        __device__ static void setup(producer_setup_args<layout> args) {
            warpgroup::decrease_registers<40>(); // decrease registers for producers
        }
        __device__ static void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                #pragma unroll
                for(int i = 0; i < 2; i++) {
                    tma::load_async(args.input.a[i], args.globals.A,
                                    {args.common.coord.x+i, args.iter}, args.inputs_arrived);
                }
                tma::load_async(args.input.b, args.globals.B,
                                {args.common.coord.y, args.iter}, args.inputs_arrived);
            }
        }
    };

    struct consumer {
        __device__ static void setup(consumer_setup_args<layout> args) {
            warpgroup::increase_registers<232>(); // increase registers for consumers
            zero(args.state.accum); 
        }
        __device__ static void compute(consumer_compute_args<layout> args) {
            warpgroup::mma_ABt(
                args.state.accum,
                args.input.a[warpgroup::groupid()],
                args.input.b
            );
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished);
        }
        __device__ static void finish(consumer_finish_args<layout> args) {
            col_vec<rt<c_dtype, 16, 128>> scale_a_rv;
            row_vec<rt<c_dtype, 16, 128>> scale_b_rv;
            warpgroup::load(scale_a_rv, args.globals.scale_a, {args.common.coord.x});
            load(scale_b_rv, args.globals.scale_b, {args.common.coord.y});
            mul_col(args.state.accum, args.state.accum, scale_b_rv);
            mul_row(args.state.accum, args.state.accum, scale_a_rv);
            warpgroup::store(args.finish.c[warpgroup::groupid()], args.state.accum);
            warpgroup::sync(warpgroup::groupid()+4);
            if(warpgroup::warpid() == 0) {
                tma::store_async(args.globals.C, args.finish.c[warpgroup::groupid()],
                                 {args.common.coord.x, args.common.coord.y});
                tma::store_async_read_wait();
            }
            if(laneid() == 0) arrive(args.finish_finished);
        }
    };
};


#include <iostream>
#include <random>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <omp.h>


template<typename mmt>
void inner_run(
    fp8e4m3 *d_A, fp8e4m3 *d_B, c_dtype *d_C, 
    c_dtype *d_scale_a, c_dtype *d_scale_b,
    size_t M, size_t N, size_t K, 
    dim3 grid, dim3 block
) {
    using a_layout = typename mmt::layout::a_layout;
    using b_layout = typename mmt::layout::b_layout;
    using c_layout = typename mmt::layout::c_layout;
    using globals  = typename mmt::layout::globals;
    a_layout Ag{d_A, nullptr, nullptr, M, K};
    b_layout Bg{d_B, nullptr, nullptr, N, K};
    c_layout Cg{d_C, nullptr, nullptr, M, N};

    // scales
    using scale_a_layout = typename mmt::layout::scale_a_layout;
    using scale_b_layout = typename mmt::layout::scale_b_layout;
    scale_a_layout scale_a{d_scale_a, nullptr, nullptr, nullptr, M};
    scale_b_layout scale_b{d_scale_b, nullptr, nullptr, nullptr, N};

    globals G{Ag, Bg, Cg, scale_a, scale_b};
    prototype::lcf::kernel<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(G);
}

void cpu_gemm(float* a, float* b, float* c, int M, int N, int K) {
    std::cout << "CPU M=" << M << " N=" << N << " K=" << K << std::endl;
    #pragma omp parallel for collapse(2) // otherwise the CPU version takes for everrrrrr
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += a[i * K + k] * b[j * K + k]; // mma_ABt
            }
            c[i * N + j] = sum;
        }
    }
}


template<typename mmt>
int run_benchmark(size_t M, size_t N, size_t K) {
    hipError_t cudaStatus;

    std::cout << "--------------------  M=" << M << " N=" << N << " K=" << K << "  --------------------\n";

    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    float *h_C_ref = new float[M * N];

    std::cout << "Allocated host memory" << std::endl;

    // Initialize random number generator
    std::random_device rd;
    std::mt19937 gen(42);
    std::normal_distribution dis(0.0f, 1.0f);

    // Initialize matrices with random values
    // for (int i = 0; i < M * K; ++i) h_A[i] = i / 100000.0f;  // dis(gen) * 0.2f; 
    // for (int i = 0; i < K * N; ++i) h_B[i] = i / 100000.0f;   // dis(gen) * 0.2f; 
    for (int i = 0; i < M * K; ++i) h_A[i] = dis(gen) * 0.2f; 
    for (int i = 0; i < K * N; ++i) h_B[i] = dis(gen) * 0.2f; 

    std::cout << "Initialized matrices" << std::endl;

    // Allocate device memory
    fp8e4m3 *d_A, *d_B;
    c_dtype *d_C;
    hipMalloc(&d_A, M*K*sizeof(fp8e4m3));
    hipMalloc(&d_B, K*N*sizeof(fp8e4m3));
    hipMalloc(&d_C, M*N*sizeof(c_dtype));
    // scales
    c_dtype *d_scale_a, *d_scale_b;
    hipMalloc(&d_scale_a, M*sizeof(c_dtype));
    hipMalloc(&d_scale_b, N*sizeof(c_dtype));

    // Check for CUDA errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Optionally, you might want to exit the program or handle the error in some way
        return -1;
    }

    std::cout << "Allocated device memory" << std::endl;

    // Perform CPU matrix multiplication for reference
    if(true) cpu_gemm(h_A, h_B, h_C_ref, M, N, K);
    std::cout << "Performed CPU matrix multiplication" << std::endl;

    //  Obtain inputs on GPU device
    const float FP8_E4M3_MAX = 448.0f;
    // const float FP8_E4M3_MIN = -448.0f;
    c_dtype *h_scale_a = new c_dtype[M];
    c_dtype *h_scale_b = new c_dtype[N];
    __hip_fp8_e4m3_fnuz *h_A_fp8_scaled = new __hip_fp8_e4m3_fnuz[M * K];
    __hip_fp8_e4m3_fnuz *h_B_fp8_scaled = new __hip_fp8_e4m3_fnuz[K * N];
    
    // row-wise scaling
    for(int row = 0; row < M; row++) {
        float max_val = 0.0f;
        for(int col = 0; col < K; col++) {
            float abs_val = std::abs(h_A[row * K + col]);
            max_val = std::max(max_val, abs_val);
        }
        h_scale_a[row] = c_dtype(max_val / FP8_E4M3_MAX); 
        if ( row < 10 ) {
            std::cout << "h_scale_a[" << row << "] = " << float(h_scale_a[row]) << ", max_val: " << max_val << std::endl;
        }
    }

    // fill h_A_fp8_scaled by following to_float8_e4m3fn. 
    for(int i = 0; i < M; i++) {
        for(int j = 0; j < K; j++) {
            h_A_fp8_scaled[i * K + j] = __hip_fp8_e4m3_fnuz(h_A[i * K + j] / float(h_scale_a[i]));
        }
    }

    // column-wise scaling
    for(int col = 0; col < N; col++) {
        float max_val = 0.0f;
        for(int row = 0; row < K; row++) {
            float abs_val = std::abs(h_B[row + col*K]);
            max_val = std::max(max_val, abs_val);
        }
        h_scale_b[col] = c_dtype(max_val / FP8_E4M3_MAX);

        if ( col < 10 ) {
            std::cout << "h_scale_b[" << col << "] = " << float(h_scale_b[col]) << ", max_val: " << max_val << std::endl;
        }
    }

    // fill h_B_fp8_scaled by following to_float8_e4m3fn
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < K; j++) {
            h_B_fp8_scaled[j + i * K] = __hip_fp8_e4m3_fnuz(h_B[j + i * K] / float(h_scale_b[i]));
        }
    }
    
    hipMemcpy(d_A, h_A_fp8_scaled, M*K*sizeof(fp8e4m3), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_fp8_scaled, K*N*sizeof(fp8e4m3), hipMemcpyHostToDevice);
    hipMemcpy(d_scale_a, h_scale_a, M*sizeof(c_dtype), hipMemcpyHostToDevice);
    hipMemcpy(d_scale_b, h_scale_b, N*sizeof(c_dtype), hipMemcpyHostToDevice);

    /* 
    Launch kernel
    */
    std::cout << "Copied matrices to device" << std::endl;
    unsigned long mem_size = MAX_SHARED_MEMORY - 1024;
    hipFuncSetAttribute(reinterpret_cast<const void*>(prototype::lcf::kernel<mmt>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

    // Launch kernel
    dim3 grid(mmt::grid(M, N, K));
    dim3 block(kittens::prototype::detail::NUM_THREADS_v<mmt>);
    std::cout << "Launching warmup kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
    for(int i = 0; i < ( 2 ); i++) { // warmup
        inner_run<mmt>(d_A, d_B, d_C, d_scale_a, d_scale_b, M, N, K, grid, block); 
    }

    // Start timing
    hipDeviceSynchronize();
    std::cout << "Launching kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
    auto start = std::chrono::high_resolution_clock::now();

    constexpr int ITERS = ( 10 );
    for(int i = 0; i < ITERS; i++) {
        inner_run<mmt>(d_A, d_B, d_C, d_scale_a, d_scale_b, M, N, K, grid, block); 
    }
    hipDeviceSynchronize();

    // End timing
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    std::chrono::duration<double> diff = end - start;
    double useconds = diff.count() * 1e6 / ITERS;

    // Calculate TFLOPs
    double flops = double(2.0) * M * N * K; // 2 FLOPs per multiply-add
    double tflops = (flops / useconds) / 1e6;

    std::cout << "Avg Kernel execution time: " << useconds << " us\n";
    std::cout << "Achieved performance: " << tflops << " TFLOPs\n";
    
    // Check for CUDA errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Optionally, you might want to exit the program or handle the error in some way
        return -1;
    }

    // Copy result back to host
    c_dtype *h_C_out = new c_dtype[M * N];
    hipMemcpy(h_C_out, d_C, M*N*sizeof(c_dtype), hipMemcpyDeviceToHost);

    std::cout << "Copied result back to host" << std::endl;

    // Convert result back to float for comparison
    for (int i = 0; i < M * N; ++i) {
        h_C[i] = float(h_C_out[i]);
    }

    std::cout << "Converted result back to float" << std::endl;

    // Check result
    float max_error = 0.0f, total_error = 0.0f, total_ref = 0.0f, total_ours=0.0f;
    float input_a = 0.0f, input_b = 0.0f;
    int error_count = 0;
    printf("Num rows: %zu, Num cols: %zu\n", M, N);
    for (int i = 0; i < M * N; ++i) {
        float error = std::abs(h_C[i] - h_C_ref[i]);
        if( error > 0.10 ) { // large because of fp8 vs fp32 numerics # error > 0.10
            if(error_count < 10) std::cout << "Error at row " << i / N << " col " << i % N << ": " << h_C[i] << " != " << h_C_ref[i] << " (ref)" << std::endl;
            else if(error_count == 700) std::cout << "Too many errors to show them all.\n";
            error_count++;
        }
        max_error = std::max(max_error, error);
        total_ref += std::abs(h_C_ref[i]);
        total_error += error;
        total_ours += std::abs(h_C[i]);
    }

    for (int i = 0; i < M * K; i++) {
        input_a += std::abs(h_A[i]);
    }
    for (int i = 0; i < K * N; i++) {
        input_b += std::abs(h_B[i]);
    }

    std::cout << std::fixed << std::setprecision(6);
    std::cout << "Max error: " << max_error << std::endl;
    std::cout << "Average error: " << total_error / M / N << std::endl;
    std::cout << "Average ref: " << total_ref / (M * N) << std::endl;
    std::cout << "Average ours: " << total_ours / M / N << std::endl;
    std::cout << "Average input_a: " << input_a / M / K << std::endl;
    std::cout << "Average input_b: " << input_b / K / N << std::endl;
    std::cout << "Error count: " << error_count << std::endl;

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_ref;
    delete[] h_C_out;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}


int main() {
    int M = 4096, N = 4096, K = 4096;
    run_benchmark<matmul_template<8>>(M, N, K);
    return 0;
}

