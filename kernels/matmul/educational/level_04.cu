#include "hip/hip_runtime.h"


#include <iostream>
#include <random>
#include <hip/hip_bf16.h>
#include <omp.h>
#include <chrono>
#include <hip/hip_runtime.h>

#include "kittens.cuh"
using namespace kittens;

constexpr int BLOCK_SIZE = 32;
#define NUM_WORKERS  (1)
#define NUM_THREADS (NUM_WORKERS*kittens::WARP_THREADS)

struct matmul_globals { 
    using sub_tile = st_bf<BLOCK_SIZE,BLOCK_SIZE>;
    using tile_gl =  gl<bf16,  1, 1, -1, -1, sub_tile>;
    tile_gl A;
    tile_gl B; 
    tile_gl C;
    int N;
};

__global__ void kernel(const __grid_constant__ matmul_globals g) {

    extern __shared__ alignment_dummy __shm[]; 
    shared_allocator al((int*)&__shm[0]);
    st_bf<BLOCK_SIZE,BLOCK_SIZE> &As = al.allocate<st_bf<BLOCK_SIZE,BLOCK_SIZE>>(); 
    st_bf<BLOCK_SIZE,BLOCK_SIZE> &Bs = al.allocate<st_bf<BLOCK_SIZE,BLOCK_SIZE>>(); 
    
    rt_bf<BLOCK_SIZE,BLOCK_SIZE> A_reg;
    rt_bf<BLOCK_SIZE,BLOCK_SIZE> B_reg;
    rt_bf<BLOCK_SIZE,BLOCK_SIZE,ducks::rt_layout::col> B_reg_col;
    rt_fl<BLOCK_SIZE,BLOCK_SIZE> C_accum;

    int col = blockIdx.x; 
    int row = blockIdx.y; 

    kittens::warp::zero(C_accum);
    int num_tiles = (g.N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    for (int tile = 0; tile < num_tiles; ++tile) {
        kittens::warp::load(As, g.A, {0, 0, row, tile});
        kittens::warp::load(Bs, g.B, {0, 0, tile, col});
        __syncthreads();
        kittens::warp::load(A_reg, As);
        kittens::warp::load(B_reg, Bs);
        kittens::warp::swap_layout(B_reg_col, B_reg);
        __syncthreads();
        kittens::warp::mma_AB(C_accum, A_reg, B_reg_col, C_accum);
        __syncthreads(); 
    }
    kittens::warp::store(g.C, C_accum, {0, 0, row, col});
}

// launch kernel
void matmul(bf16* A, bf16* B, bf16* C, size_t N) { 

    // global pointers
    using a_gl = matmul_globals::tile_gl;
    using b_gl = matmul_globals::tile_gl; 
    using c_gl = matmul_globals::tile_gl;
    a_gl  a_arg{A, nullptr, nullptr, N, N};
    b_gl  b_arg{B, nullptr, nullptr, N, N};
    c_gl  c_arg{C, nullptr, nullptr, N, N};
    matmul_globals g{a_arg, b_arg, c_arg, (int)N}; 

    // launch
    dim3 blocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);  // Watch out for requesting too many!
    unsigned long mem_size = 100000;
    hipDeviceSynchronize();
    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
    kernel<<<blocks, NUM_THREADS, mem_size>>>(g);
    CHECK_CUDA_ERROR(hipGetLastError());
    hipDeviceSynchronize();
}

#include "launch.cu"
