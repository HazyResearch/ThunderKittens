#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda/atomic>
#include <iostream>
#include <cassert>
#include <vector>
#include <chrono>

#include "kittens.cuh"

const int NUM_DEVICES = 8;

using namespace kittens;

// CUDA driver API
#define CUCHECK(cmd) do {                                     \
    hipError_t err = cmd;                                       \
    if (err != hipSuccess) {                                \
        const char *errStr;                                   \
        hipDrvGetErrorString(err, &errStr);                       \
        fprintf(stderr, "Failed: CUDA error %s:%d '%s'\n",    \
            __FILE__, __LINE__, errStr);                      \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(0)

// CUDA runtime API
#define CUDACHECK(cmd) do {                                   \
    hipError_t err = cmd;                                    \
    if (err != hipSuccess) {                                 \
        fprintf(stderr, "Failed: CUDA error %s:%d '%s'\n",    \
            __FILE__, __LINE__, hipGetErrorString(err));     \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(0)

template <ducks::sync_manager::all SyncManager>
__global__ void test_barrier_kernel(SyncManager sm, int sync_id, int dev_id) {
    using gang = kittens::gang<4>;
    gang::sync<sync_level::GRID>(sm, sync_id, dev_id); 
}

int main() {
    assert(NUM_DEVICES > 1); 

    CUCHECK(hipInit(0)); 
    
    std::vector<hipStream_t> streams(NUM_DEVICES);
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUDACHECK(hipStreamCreate(&streams[dev_idx]));
    }

    /*
    Run kernel to profile barrier 
    */
    constexpr int num_blocks = 1000;
    dim3 grid(num_blocks, 1, 1);
    dim3 block(256, 1, 1);
    
    int device_ids[NUM_DEVICES];
    for (int i = 0; i < NUM_DEVICES; ++i) device_ids[i] = i;
    CUDACHECK(hipSetDevice(0));
    KittensClub club(device_ids, NUM_DEVICES);
    
    const int PROFILE_ITERS = 50;

    sync_manager sm = sync_manager<NUM_DEVICES, sync_level::GRID, 16, num_blocks>::create(device_ids);

    auto start = std::chrono::high_resolution_clock::now();
    for (int iter = 0; iter < PROFILE_ITERS; iter++) {
        club.execute([&](int dev_idx) {
            test_barrier_kernel<<<grid, block, 0, streams[dev_idx]>>>(sm, 0, dev_idx);
            CUDACHECK(hipStreamSynchronize(streams[dev_idx]));
        });
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    double avg_time_ms = (elapsed.count() * 1e3) / PROFILE_ITERS;

    std::cout << "Effective barrier overhead (ms): " << avg_time_ms << std::endl;

    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        CUDACHECK(hipSetDevice(dev_idx));
        CUDACHECK(hipStreamDestroy(streams[dev_idx]));
    }
    sm.free();

    return 0;
}
