#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

/*

consumer_state.att_block in a warp looks like this:

[ 
  x x x x x x ... 192 or 128 columns ... x x x x x x
  ... 16 rows of it
  x x x x x x ... 192 or 128 columns ... x x x x x x
]

Logically in a warpgroup, it looks like this:
[
  x x x x x x ... 192 or 128 columns ... x x x x x x
  ... 64 rows of it
  x x x x x x ... 192 or 128 columns ... x x x x x x
]

*/

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;
template<int D, int NUM_WORKERS> struct attn_fwd_layout {
    using qo_tile   = st_bf<64, D>;
    using kv_tile   = st_bf<D==64?192:128, D>;
    using qo_global = kittens::gl<bf16, -1, -1, -1, D, qo_tile>;
    using kv_global = kittens::gl<bf16, -1, -1, -1, D, kv_tile>;
    struct globals {
        qo_global O, Q;
        kv_global K, V;
        int dynamic_shared_memory() { return MAX_SHARED_MEMORY - 2000; }
        dim3 grid()  { return dim3(132); }
        dim3 block() { return dim3((12+4)*WARP_THREADS); }
    };
    struct input_block    { kv_tile k, v; };
    struct scratch_block  { qo_tile q[NUM_WORKERS]; };
    struct common_state   { int batch, head, seq; };
    struct consumer_state {
        rt_fl<16, qo_tile::cols> o_reg;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec, norm_vec;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec_last_scaled, max_vec_scaled;
        rt_fl<16, kv_tile::rows> att_block;
        rt_bf<16, kv_tile::rows> att_block_mma;
    };
};
template<int D, bool causal> struct attn_fwd_template {
    static constexpr int NUM_CONSUMER_WARPS = 12, NUM_WORKERS = NUM_CONSUMER_WARPS/4, INPUT_PIPE_STAGES = 2;
    using layout = attn_fwd_layout<D, NUM_WORKERS>;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int task_id = gridDim.x*args.task_iter + blockIdx.x;
        int seq_q = (args.globals.Q.rows + NUM_WORKERS*layout::qo_tile::rows - 1)/(NUM_WORKERS*layout::qo_tile::rows);
        args.common.batch = task_id / (seq_q*args.globals.K.depth); task_id -= args.common.batch * seq_q * args.globals.K.depth;
        args.common.head  = task_id / seq_q;                        task_id -= args.common.head  * seq_q;
        args.common.seq   = task_id;
        args.num_iters = args.common.batch < args.globals.Q.batch ? (args.globals.K.rows + layout::kv_tile::rows - 1)/(layout::kv_tile::rows) : -1;
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {
            warpgroup::producer_registers();
        }
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                tma::load_async(args.input.k, args.globals.K, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
                tma::load_async(args.input.v, args.globals.V, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
            }
            else if(laneid() == 0) arrive(args.inputs_arrived);
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            warpgroup::consumer_registers<NUM_WORKERS>();
            if((args.common.seq*NUM_WORKERS + warpgroup::groupid())*layout::qo_tile::rows < args.globals.Q.rows) // out of bounds?
                warpgroup::load(args.scratch.q[warpgroup::groupid()], args.globals.Q,
                                {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
            zero(args.state.o_reg);
            zero(args.state.norm_vec);
            neg_infty(args.state.max_vec);
            warpgroup::sync(warpgroup::groupid());
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            int qidx = (args.common.seq*NUM_WORKERS+warpgroup::groupid())*layout::qo_tile::rows;
            int kvidx = args.iter*layout::kv_tile::rows;
            /*
            How to know whether to skip a whole block if it's causal?
            Q index is (args.common.seq*NUM_WORKERS+warpgroup::groupid()) * layout::qo_tile::rows
            K and V index is (args.iter*layout::kv_tile::rows) * args.globals.K.depth

            We can skip this block if:
            * causal is True and K/V index is greater than Q index
            * which happens when args.iter*layout::kv_tile::rows > args.common.seq*NUM_WORKERS+warpgroup::groupid()
            */
            if (!causal || kvidx <= qidx) {
                constexpr float TEMPERATURE_SCALE = (D == 128) ? 0.08838834764f*1.44269504089f : 0.125f*1.44269504089f;
                // A = Q @ K.T
                warpgroup::mm_ABt(args.state.att_block, args.scratch.q[warpgroup::groupid()], args.input.k);
                mul(args.state.max_vec_last_scaled, args.state.max_vec, TEMPERATURE_SCALE);
                warpgroup::mma_async_wait();
                // softmax
                right_fill(args.state.att_block, args.state.att_block, args.globals.K.rows - args.iter*layout::kv_tile::rows, base_types::constants<float>::neg_infty());
                if (causal) {
                    // if qidx - kvidx is less than the number of columns, this tile passes the diagonal
                    if (qidx - kvidx < layout::qo_tile::cols) {
                        // blocks are wider than they are tall, so we have mulitple blocks on the diagonal
                        // kvidx - qidx gives the (negative) offset based on block
                        // 16 * (warpgroup::warpid() % 4) gives the offset based on the warp
                        tril(args.state.att_block, args.state.att_block, kvidx - qidx - 16 * (warpgroup::warpid() % 4), base_types::constants<float>::neg_infty());
                    }
                }
                row_max(args.state.max_vec, args.state.att_block, args.state.max_vec); // accumulate onto the max_vec
                mul(args.state.max_vec_scaled, args.state.max_vec, TEMPERATURE_SCALE);
                mul(args.state.att_block, args.state.att_block, TEMPERATURE_SCALE);
                sub_row(args.state.att_block, args.state.att_block, args.state.max_vec_scaled);
                exp2(args.state.att_block, args.state.att_block);
                sub(args.state.max_vec_last_scaled, args.state.max_vec_last_scaled, args.state.max_vec_scaled);
                exp2(args.state.max_vec_last_scaled, args.state.max_vec_last_scaled);
                mul(args.state.norm_vec, args.state.norm_vec, args.state.max_vec_last_scaled);
                row_sum(args.state.norm_vec, args.state.att_block, args.state.norm_vec); // accumulate onto the norm_vec
                mul_row(args.state.o_reg, args.state.o_reg, args.state.max_vec_last_scaled); // normalize o_reg before mma
                copy(args.state.att_block_mma, args.state.att_block); // convert to bf16 for mma
                // O += A @ V
                warpgroup::mma_AB(args.state.o_reg, args.state.att_block_mma, args.input.v);
                warpgroup::mma_async_wait();
            }
            if(laneid() == 0) arrive(args.inputs_finished); // done!
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            if((args.common.seq*NUM_WORKERS+warpgroup::groupid())*64 < args.globals.Q.rows) { // out of bounds?
                div_row(args.state.o_reg, args.state.o_reg, args.state.norm_vec);
                auto &o_smem = reinterpret_cast<typename layout::qo_tile&>(args.scratch.q[warpgroup::groupid()]);
                warpgroup::store(o_smem, args.state.o_reg);
                warpgroup::sync(warpgroup::groupid());
                if(warpgroup::warpid() == 0)
                    tma::store_async(args.globals.O, o_smem, {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
                tma::store_async_read_wait();
            }
            __syncwarp();
            if(laneid() == 0) arrive(args.finish_finished); // done!
        }
    };
};
// kernel is kittens::prototype::lcf::kernel<attn_fwd_template<HEAD_DIM>>;

PYBIND11_MODULE(attn_fwd_pybind, m) {
    m.doc() = "TK Attention Forward Demo";
    py::bind_kernel<kittens::prototype::lcf::kernel<attn_fwd_template<128, false>>>(m, "fwd_128_noncausal",
        &attn_fwd_layout<128, 3>::globals::O,
        &attn_fwd_layout<128, 3>::globals::Q,
        &attn_fwd_layout<128, 3>::globals::K,
        &attn_fwd_layout<128, 3>::globals::V
    );
    py::bind_kernel<kittens::prototype::lcf::kernel<attn_fwd_template<128, true>>>(m, "fwd_128_causal",
        &attn_fwd_layout<128, 3>::globals::O,
        &attn_fwd_layout<128, 3>::globals::Q,
        &attn_fwd_layout<128, 3>::globals::K,
        &attn_fwd_layout<128, 3>::globals::V
    );
}