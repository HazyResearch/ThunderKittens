#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;
template<int D, int NUM_WORKERS> struct attn_fwd_layout {
    using qo_tile   = st_bf<64, D>;
    using kv_tile   = st_bf<D==64?192:128, D>;
    using qo_global = kittens::gl<bf16, -1, -1, -1, D, qo_tile>;
    using kv_global = kittens::gl<bf16, -1, -1, -1, D, kv_tile>;
    struct globals { qo_global O, Q; kv_global K, V; };
    struct input_block    { kv_tile k, v; };
    struct scratch_block  { qo_tile q[NUM_WORKERS]; };
    struct common_state   { int batch, head, seq; };
    struct consumer_state {
        rt_fl<16, qo_tile::cols> o_reg;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec, norm_vec;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec_last_scaled, max_vec_scaled;
        rt_fl<16, kv_tile::rows> att_block;
        rt_bf<16, kv_tile::rows> att_block_mma;
    };
};
template<int D> struct attn_fwd_template {
    static constexpr int NUM_CONSUMER_WARPS = 12, NUM_WORKERS = NUM_CONSUMER_WARPS/4, INPUT_PIPE_STAGES = 2;
    using layout = attn_fwd_layout<D, NUM_WORKERS>;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int task_id = gridDim.x*args.task_iter + blockIdx.x;
        int seq_q = (args.globals.Q.rows() + NUM_WORKERS*layout::qo_tile::rows - 1)/(NUM_WORKERS*layout::qo_tile::rows);
        args.common.batch = task_id / (seq_q*args.globals.K.depth()); task_id -= args.common.batch * seq_q * args.globals.K.depth();
        args.common.head  = task_id / seq_q;                        task_id -= args.common.head  * seq_q;
        args.common.seq   = task_id;
        args.num_iters = args.common.batch < args.globals.Q.batch() ? (args.globals.K.rows() + layout::kv_tile::rows - 1)/(layout::kv_tile::rows) : -1;
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {
            warpgroup::producer_registers();
        }
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                tma::load_async(args.input.k, args.globals.K, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
                tma::load_async(args.input.v, args.globals.V, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
            }
            else if(laneid() == 0) arrive(args.inputs_arrived);
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            warpgroup::consumer_registers<NUM_WORKERS>();
            if((args.common.seq*NUM_WORKERS + warpgroup::groupid())*layout::qo_tile::rows < args.globals.Q.rows()) // out of bounds?
                warpgroup::load(args.scratch.q[warpgroup::groupid()], args.globals.Q,
                                {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
            args.state.o_reg = 0.f;
            args.state.norm_vec = 0.f;
            args.state.max_vec = base_types::constants<float>::neg_infty();
            warpgroup::sync(warpgroup::groupid());
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            constexpr float TEMPERATURE_SCALE = (D == 128) ? 0.08838834764f*1.44269504089f : 0.125f*1.44269504089f;
            // A = Q @ K.T
            warpgroup::mm<transpose::N, transpose::T>(args.state.att_block, args.scratch.q[warpgroup::groupid()], args.input.k);
            args.state.max_vec_last_scaled = args.state.max_vec * TEMPERATURE_SCALE;
            warpgroup::mma_async_wait();
            // softmax
            right_fill(args.state.att_block, args.state.att_block, args.globals.K.rows() - args.iter*layout::kv_tile::rows, base_types::constants<float>::neg_infty());
            args.state.max_vec = max<axis::COL>(args.state.att_block, args.state.max_vec); // accumulate onto the max_vec
            args.state.max_vec_scaled = args.state.max_vec * TEMPERATURE_SCALE;
            args.state.att_block = exp2((args.state.att_block*TEMPERATURE_SCALE) - args.state.max_vec_scaled);
            args.state.max_vec_last_scaled = exp2(args.state.max_vec_last_scaled - args.state.max_vec_scaled);
            args.state.norm_vec *= args.state.max_vec_last_scaled;
            args.state.norm_vec = sum<axis::COL>(args.state.att_block, args.state.norm_vec); // accumulate onto the norm_vec
            args.state.o_reg *= args.state.max_vec_last_scaled; // normalize o_reg before mma
            args.state.att_block_mma = args.state.att_block; // convert to bf16 for mma
            // O += A @ V
            warpgroup::mma<transpose::N, transpose::N>(args.state.o_reg, args.state.att_block_mma, args.input.v);
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.inputs_finished); // done!
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            if((args.common.seq*NUM_WORKERS+warpgroup::groupid())*64 < args.globals.Q.rows()) { // out of bounds?
                args.state.o_reg /= args.state.norm_vec;
                auto &o_smem = reinterpret_cast<typename layout::qo_tile&>(args.scratch.q[warpgroup::groupid()]);
                warpgroup::store(o_smem, args.state.o_reg);
                warpgroup::sync(warpgroup::groupid());
                if(warpgroup::warpid() == 0)
                    tma::store_async(args.globals.O, o_smem, {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
                tma::store_async_read_wait();
            }
            __syncwarp();
            if(laneid() == 0) arrive(args.finish_finished); // done!
        }
    };
};
// kernel is kittens::prototype::lcf::kernel<attn_fwd_template<HEAD_DIM>>;

#include "h100_lcf_harness.impl"