#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"

/*

consumer_state.att_block in a warp looks like this:

[ 
  x x x x x x ... 192 or 128 columns ... x x x x x x
  ... 16 rows of it
  x x x x x x ... 192 or 128 columns ... x x x x x x
]

Logically in a warpgroup, it looks like this:
[
  x x x x x x ... 192 or 128 columns ... x x x x x x
  ... 64 rows of it
  x x x x x x ... 192 or 128 columns ... x x x x x x
]

*/

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcf;
template<int D, int NUM_WORKERS> struct attn_fwd_layout {
    using qo_tile   = st_bf<64, D>;
    using kv_tile   = st_bf<D==64?192:128, D>;
    using qo_global = kittens::gl<bf16, -1, -1, -1, D, qo_tile>;
    using kv_global = kittens::gl<bf16, -1, -1, -1, D, kv_tile>;
    struct globals { qo_global O, Q; kv_global K, V; };
    struct input_block    { kv_tile k, v; };
    struct scratch_block  { qo_tile q[NUM_WORKERS]; };
    struct common_state   { int batch, head, seq; };
    struct consumer_state {
        rt_fl<16, qo_tile::cols> o_reg;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec, norm_vec;
        col_vec<rt_fl<16, kv_tile::rows>> max_vec_last_scaled, max_vec_scaled;
        rt_fl<16, kv_tile::rows> att_block;
        rt_bf<16, kv_tile::rows> att_block_mma;
    };
};
template<int D, bool causal> struct attn_fwd_template {
    static constexpr int NUM_CONSUMER_WARPS = 12, NUM_WORKERS = NUM_CONSUMER_WARPS/4, INPUT_PIPE_STAGES = 2;
    using layout = attn_fwd_layout<D, NUM_WORKERS>;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int task_id = gridDim.x*args.task_iter + blockIdx.x;
        int seq_q = (args.globals.Q.rows + NUM_WORKERS*layout::qo_tile::rows - 1)/(NUM_WORKERS*layout::qo_tile::rows);
        args.common.batch = task_id / (seq_q*args.globals.K.depth); task_id -= args.common.batch * seq_q * args.globals.K.depth;
        args.common.head  = task_id / seq_q;                        task_id -= args.common.head  * seq_q;
        args.common.seq   = task_id;
        args.num_iters = args.common.batch < args.globals.Q.batch ? (args.globals.K.rows + layout::kv_tile::rows - 1)/(layout::kv_tile::rows) : -1;
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {
            warpgroup::producer_registers();
        }
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input);
                tma::load_async(args.input.k, args.globals.K, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
                tma::load_async(args.input.v, args.globals.V, {args.common.batch, args.common.head, args.iter, 0}, args.inputs_arrived);
            }
            else if(laneid() == 0) arrive(args.inputs_arrived);
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            warpgroup::consumer_registers<NUM_WORKERS>();
            if((args.common.seq*NUM_WORKERS + warpgroup::groupid())*layout::qo_tile::rows < args.globals.Q.rows) // out of bounds?
                warpgroup::load(args.scratch.q[warpgroup::groupid()], args.globals.Q,
                                {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
            zero(args.state.o_reg);
            zero(args.state.norm_vec);
            neg_infty(args.state.max_vec);
            warpgroup::sync(warpgroup::groupid());
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            /*
            How to know whether to skip a whole block if it's causal?
            Q index is (args.common.seq*NUM_WORKERS+warpgroup::groupid()) * layout::qo_tile::rows
            K and V index is (args.iter*layout::kv_tile::rows) * args.globals.K.depth

            We can skip this block if:
            * causal is True and K/V index is greater than Q index
            * which happens when args.iter*layout::kv_tile::rows > args.common.seq*NUM_WORKERS+warpgroup::groupid()
            */
            if (!causal || (causal && args.iter*layout::kv_tile::rows > args.common.seq*NUM_WORKERS+warpgroup::groupid())) {
                if (warpgroup::laneid() == 0) {
                    printf("warpgroup_id: %d, iter: %d, seq: %d, kv_idx: %d, q_idx: %d\n", warpgroup::groupid(), args.iter, args.common.seq, args.iter*layout::kv_tile::rows, args.common.seq*NUM_WORKERS+warpgroup::groupid());
                }
                constexpr float TEMPERATURE_SCALE = (D == 128) ? 0.08838834764f*1.44269504089f : 0.125f*1.44269504089f;
                // A = Q @ K.T
                warpgroup::mm_ABt(args.state.att_block, args.scratch.q[warpgroup::groupid()], args.input.k);
                mul(args.state.max_vec_last_scaled, args.state.max_vec, TEMPERATURE_SCALE);
                warpgroup::mma_async_wait();
                // softmax
                right_fill(args.state.att_block, args.state.att_block, args.globals.K.rows - args.iter*layout::kv_tile::rows, base_types::constants<float>::neg_infty());
                if (causal) {
                    // we need to do a causal mask here

                }
                row_max(args.state.max_vec, args.state.att_block, args.state.max_vec); // accumulate onto the max_vec
                mul(args.state.max_vec_scaled, args.state.max_vec, TEMPERATURE_SCALE);
                mul(args.state.att_block, args.state.att_block, TEMPERATURE_SCALE);
                sub_row(args.state.att_block, args.state.att_block, args.state.max_vec_scaled);
                exp2(args.state.att_block, args.state.att_block);
                sub(args.state.max_vec_last_scaled, args.state.max_vec_last_scaled, args.state.max_vec_scaled);
                exp2(args.state.max_vec_last_scaled, args.state.max_vec_last_scaled);
                mul(args.state.norm_vec, args.state.norm_vec, args.state.max_vec_last_scaled);
                row_sum(args.state.norm_vec, args.state.att_block, args.state.norm_vec); // accumulate onto the norm_vec
                mul_row(args.state.o_reg, args.state.o_reg, args.state.max_vec_last_scaled); // normalize o_reg before mma
                copy(args.state.att_block_mma, args.state.att_block); // convert to bf16 for mma
                // O += A @ V
                warpgroup::mma_AB(args.state.o_reg, args.state.att_block_mma, args.input.v);
                warpgroup::mma_async_wait();
            }
            if(laneid() == 0) arrive(args.inputs_finished); // done!
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            if((args.common.seq*NUM_WORKERS+warpgroup::groupid())*64 < args.globals.Q.rows) { // out of bounds?
                div_row(args.state.o_reg, args.state.o_reg, args.state.norm_vec);
                auto &o_smem = reinterpret_cast<typename layout::qo_tile&>(args.scratch.q[warpgroup::groupid()]);
                warpgroup::store(o_smem, args.state.o_reg);
                warpgroup::sync(warpgroup::groupid());
                if(warpgroup::warpid() == 0)
                    tma::store_async(args.globals.O, o_smem, {args.common.batch, args.common.head, args.common.seq*NUM_WORKERS+warpgroup::groupid(), 0});
                tma::store_async_read_wait();
            }
            __syncwarp();
            if(laneid() == 0) arrive(args.finish_finished); // done!
        }
    };
};
// kernel is kittens::prototype::lcf::kernel<attn_fwd_template<HEAD_DIM>>;

#include "h100_lcf_harness.impl"