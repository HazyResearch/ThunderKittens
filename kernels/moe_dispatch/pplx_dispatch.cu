#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvtx3/nvToolsExt.h>

#include "all_to_all/internode.h"
#include "core/device_utils.h"
#include "core/nvshmem_utils.h"
#include "core/utils.h"

using namespace pplx;

template <unsigned NUM_WARPS, bool DO_SEND, bool DO_RECV>
__global__ __launch_bounds__(NUM_WARPS * 32, 1) void dispatchKernel(
    int32_t *outNumTokensPerExpert,
    size_t outNumTokensPerExpertStrideElem,
    std::byte *expertX, // destination array for received tokens for a rank for token data
    size_t expertXStrideElem,
    size_t expertXStrideRow,
    std::byte *expertXScale, // destination array for received tokens for a rank for token scale factors
    size_t expertXScaleStrideElem,
    size_t expertXScaleStrideRow,
    std::byte *dpX, // source array to get tokens for a rank's token data to send off 
    size_t dpXStrideElem, // hiddenDim * sizeof() ==> Allows to access start of token data
    std::byte *dpXScale, // source array to get tokens for a rank's token scale factors to send off
    size_t dpXScaleStrideElem,
    uint32_t *indices, // indices is per device, routing information for the "m" tokens sending
    size_t indicesStrideElem,
    size_t indicesStrideRow,
    size_t maxNumTokens,
    size_t numExperts, // 256 experts
    unsigned rank,
    unsigned worldSize,
    unsigned dpSize,
    size_t hiddenDim,
    size_t hiddenDimScale,
    size_t numExpertsPerToken, // 8 experts
    unsigned *boundM,
    unsigned m,
    uint32_t *numTokensPerDP,
    uint32_t *sourceExpert,
    uint32_t *sourceIndex,
    uint32_t *sourceOffset,
    uint32_t *sourceGroup,

    // Buffer that holds outgoing token count signals
    uint64_t *numTokensBuffer, // SIZE: numLocalExperts * numDPGroups
    // Buffer that holds received token count signals
    uint64_t *numRecvBuffer, // SIZE: numLocalExperts * numDPGroups
    // Symmetric input buffer used to pack tokens before sending
    std::byte *xBufferIn, // SIZE: maxNumTokens * perTokenBytes
    // Output buffer where incoming tokens are written
    std::byte *xBufferOut // SIZE: maxBatchTokens * perTokenBytes
) {
  // Determine the rank, DP rank and per-rank constants.
  const unsigned numLocalExperts = numExperts / worldSize; // 256 / 8 => 32
  const unsigned numDPGroups = worldSize / dpSize; // 8 / 1 => 8
  const unsigned dpGroup = rank / dpSize; // dpGroup = rank
  const unsigned dpRank = rank % dpSize; // dpRank = 0
  const unsigned tokenDim = hiddenDim + hiddenDimScale;
  const unsigned tokenStride =
      device::round_up<unsigned>(tokenDim + sizeof(uint32_t), sizeof(int4));
  const unsigned WARP_SIZE = 32;
  const unsigned warpId = threadIdx.x / WARP_SIZE;
  const unsigned laneId = threadIdx.x % WARP_SIZE;

  // Determine the number of tokens populated which are to be sent.
  const unsigned numTokens = boundM ? __ldg(boundM) : m;
  ROSE_DEVICE_ASSERT(numTokens <= maxNumTokens);
  ROSE_DEVICE_ASSERT(
      hiddenDimScale == 0 || numTokens == 0 || (expertXScale != nullptr && dpXScale != nullptr)
  );

  // Zero out the shared memory buffer.
  if constexpr (DO_SEND) {
    // Each tokenIndex[expert] will hold the number of tokens processed for that expert
    extern __shared__ uint32_t tokenIndex[];
    for (uint32_t i = threadIdx.x; i < numExperts; i += blockDim.x) {
      tokenIndex[i] = 0;
    }
    __syncthreads();

    if (warpId + 1 == NUM_WARPS) { // If last warp
      // Each block is responsible for some subset of experts... 
      // The loop then iterates over destination experts that are assigned to this block
      // The loop jumps by gridDim.x * x to cover all experts in round robin fashion
      for (unsigned dstExpert = blockIdx.x * dpSize + dpRank; dstExpert < numExperts;
           dstExpert += gridDim.x * dpSize) {
        const uint32_t dstRank = dstExpert / numLocalExperts; // dstGPU 
        const uint32_t dstLocalExpert = dstExpert % numLocalExperts; // dstLocalExpert

        unsigned count = 0;

        // Each thread in the warp processes a subset of the total tokens for the current expert
        // The indices array contains the expert assignments for each token
        #pragma unroll
        for (uint32_t i = laneId; i < numTokens * numExpertsPerToken; i += WARP_SIZE) {
          unsigned expert = __ldg(&indices[i]);
          if (expert == dstExpert) {
            count += 1;
          }
        }

        // Perform warp level reduction to get total token count for this expert
        unsigned numTokensPerExpert = device::warp_sum(count);
        uint64_t *dstCount = &numTokensBuffer[dstLocalExpert * numDPGroups + dpGroup];

        if (laneId == 0) {
          // send +1 here such that if numTokensPerExpert is 0, we can still signal
          nvshmemx_signal_op(dstCount, numTokensPerExpert + 1, NVSHMEM_SIGNAL_SET, dstRank);
        }
      }

      // Clear out some buffers.
      // OPTIONAL: Ensure per-expert output count starts at 0
      if (blockIdx.x == 0) {
        for (uint32_t i = laneId; i < numLocalExperts; i += WARP_SIZE) {
          outNumTokensPerExpert[i] = 0;
        }
      }
    } else {
      // Send the tokens to the destination ranks through RDMA.
      const unsigned numGroupWarps = NUM_WARPS - 1;
      const unsigned numGroupThreads = numGroupWarps * WARP_SIZE;
      
      // numTokens here is "m", which is random number between 0 and MAX_NUM_TOKENS
      for (unsigned i = 0; i < numTokens; i++) {
        // If the token is assigned to this block, handle it.
        if (i % (gridDim.x * dpSize) == (blockIdx.x * dpSize + dpRank)) {
          // Copy the token to the symmetric buffer.
          // xBufferIn is symmetric input buffer where tokens are staged before sending 
          // Needs to pack into xInPtr as nvshmem ops require both sides to be symmetric memory
          std::byte *xInPtr = xBufferIn + i * tokenStride;
          const int4 *srcX = (int4 *)(dpX + i * dpXStrideElem);
          const int4 *srcXScale = (int4 *)(dpXScale + i * dpXScaleStrideElem);
          
          // Each thread in warp group (NUM_WARPS - 1) copies a slice of tokens data
          // Loop through token's data in units of int4 until complete tokenDim is copied 
          // "tokenDim" here is "hiddenDim + hiddenDimScale"
          for (unsigned d = threadIdx.x; d * sizeof(int4) < tokenDim; d += numGroupThreads) {
            if (d * sizeof(int4) < hiddenDim) {
              ((int4 *)xInPtr)[d] = srcX[d];
            } else {
              ((int4 *)xInPtr)[d] = srcXScale[d - hiddenDim / sizeof(int4)];
            }
          }
          
          // Have one thread (thread 0) store the token index in the symmetric buffer.
          // This index is later used to track where the token originated.
          if (threadIdx.x == 0) {
            *((uint32_t *)(xInPtr + tokenDim)) = i;
          }


          // Synchronize the warps within this warp group => Ensure copy is completed
          asm volatile("bar.sync 1, %0;" ::"r"(numGroupThreads));

          // Send the token to the other ranks, one send per warp.
          // Each warp in the warp group sends the token based on a subset of destination assignments
          for (unsigned j = warpId; j < numExpertsPerToken; j += numGroupWarps) {
            // Load destionation expert for this token 
            const uint32_t dstExpert = __ldg(&indices[i * numExpertsPerToken + j]);
            // Compute which GPU the token is being sent to
            const uint32_t dstRank = dstExpert / numLocalExperts;
            const uint32_t dstLocalExpert = dstExpert % numLocalExperts;

            // Retrieve current count of tokens sent to this expert
            const uint32_t index = tokenIndex[dstExpert];
            // group = dstLocalExpert * 1 + rank
            const uint32_t group = dstLocalExpert * numDPGroups + dpGroup;
            // Compute destination slot within remote's buffer
            // Each group has a fixed capacity (maxNumTokens) 
            const unsigned loc = group * maxNumTokens + index;

            std::byte *destPointer = xBufferOut + loc * tokenStride;
            nvshmemx_putmem_signal_nbi_warp(
                destPointer,
                xInPtr,
                tokenStride,
                &numRecvBuffer[group],
                1,
                NVSHMEM_SIGNAL_ADD,
                dstRank // Where token should be sent
            );
          }
        }

        // Replicate the token count calculation across all blocks.
        // Need to update this such that for future tokens, can index into correct location
        // in xBufferOut
        if (warpId == 0 && laneId < numExpertsPerToken) {
          uint32_t dstExpert = __ldg(&indices[i * numExpertsPerToken + laneId]);
          tokenIndex[dstExpert]++;
        }
      }
    }
  }

  if constexpr (DO_RECV) {
    // Each thread in the grid processes one (expert, DP group) pair
    // The total number of pairs is numLocalExperts * numDPGroups: (32 * 1) => Only first 32 threads get inside loop
    /*
    expertAndGroup = global thread index for grid
    numLocalExperts * numDPGroups = 32 * 1 = 32 => only first 32 threads in entire grid get inside loop
    blockDim.x * gridDim.x = total number of threads in the grid
    */
    for (unsigned expertAndGroup = blockIdx.x * blockDim.x + threadIdx.x;
         expertAndGroup < numLocalExperts * numDPGroups;
         expertAndGroup += blockDim.x * gridDim.x) {
      // 
      const uint32_t srcDpGroup = expertAndGroup % numDPGroups; // srcDpGroup = 0
      const uint32_t srcLocalExpert = expertAndGroup / numDPGroups; // srcLocalExpert = expertAndGroup / 1
      const size_t slot = srcLocalExpert * numDPGroups + srcDpGroup; // slot = expertAndGroup * 8

      // Fetch the token count per DP, which is non-zero to indicate receipt.
      // Afterwards, wait for exactly that many tokens to be sent to us.
      nvshmem_uint64_wait_until(&numTokensBuffer[slot], NVSHMEM_CMP_NE, 0);
      // numTokens here is updated value after corresponding warp has finished processing 
      // how many tokens should expect to receive 
      size_t numTokens = numTokensBuffer[slot] - 1;
      nvshmem_uint64_wait_until(&numRecvBuffer[slot], NVSHMEM_CMP_EQ, numTokens);

      // Store the token count locally.
      numTokensPerDP[slot] = numTokens;
      atomicAdd(&outNumTokensPerExpert[srcLocalExpert], numTokens);

      // Clean the buffers.
      numTokensBuffer[slot] = 0;
      numRecvBuffer[slot] = 0;
    }
    // SYNC ENTIRE GRID!!!
    cg::this_grid().sync();

    /*
    After sync: Now need to copy tokens from symmetric shared buffer to local output buffer
    */
    // Copy the tokens from the symmetric buffer to the output buffer.
    unsigned expert = 0;
    unsigned dp = 0;
    unsigned offset = 0;
    unsigned start = 0;
    // maxBatchTokens: 8 * 1 * 128 = 1024 
    const uint32_t maxBatchTokens = numLocalExperts * numDPGroups * maxNumTokens;
    /*
    Each block handles a seperate token
    */
    for (uint32_t token = blockIdx.x; token < maxBatchTokens; token += gridDim.x) {
      // Find the expert, DP group and index for this token.
      unsigned j = token - offset;

      /*
      After receiver phase has finished waiting, kernel now needs to determine how to place token
      into correct output slot
        - All tokens are in xBufferOut but can belong to different experts and distributed across diff DP groups
        - tokens in the buffer are store sequentially => packed one after other regardless of expert belong to 
      
      While loop helps maps the global token index to which expert should get this token and what is tokens idx
      with that expert DP group

      Performs a prefix sum lookup over numTokensPerDp array, subtracting counts until it finds
      correct bucket for current token idx

      numTokensPerDP here is number of tokens that should be received for current expert
      */
      while (offset + __ldg(&numTokensPerDP[expert * numDPGroups + dp]) <= token) {
        // Since current token not in current group, add number of tokens in group to offset
        offset += __ldg(&numTokensPerDP[expert * numDPGroups + dp]);
        j = token - offset;
        if (++dp == numDPGroups) {
          dp = 0;
          start = offset;
          if (++expert == numLocalExperts) {
            break;
          }
        }
      }
      if (expert >= numLocalExperts) {
        break;
      }

      // Copy the token to the output buffer.
      const uint32_t group = expert * numDPGroups + dp;
      const std::byte *xTokenBuffer =
          (const std::byte *)xBufferOut + (group * maxNumTokens + j) * tokenStride;

      const unsigned loc = token - start;
      const int4 *srcX = (int4 *)xTokenBuffer;

      // expertX here is where to copy local received tokens to
      int4 *dstX = (int4 *)(expertX + expert * expertXStrideRow + loc * expertXStrideElem);
      int4 *dstXScale =
          (int4 *)(expertXScale + expert * expertXScaleStrideRow + loc * expertXScaleStrideElem);
      for (unsigned k = threadIdx.x; k * sizeof(int4) < tokenDim; k += blockDim.x) {
        if (k * sizeof(int4) < hiddenDim) {
          dstX[k] = srcX[k];
        } else {
          dstXScale[k - hiddenDim / sizeof(int4)] = srcX[k];
        }
      }

      if (threadIdx.x == 0) {
        sourceIndex[token] = *((uint32_t *)(xTokenBuffer + tokenDim)); // OG token index 
        sourceExpert[token] = expert + 1; // Save expert ID 
        sourceOffset[token] = loc;
        sourceGroup[token] = dp;
      }
    }
  }
}

void AllToAllInterNode::dispatch(
    const Strided1D<int32_t> &outNumTokensPerExpert,
    const Strided2D<std::byte> &expertX,
    const Strided2D<std::byte> &expertXScale,
    const Strided1D<std::byte> &dpX,
    const Strided1D<std::byte> &dpXScale,
    const Strided2D<uint32_t> &indices,
    unsigned m,
    const unsigned *boundM,
    SplitMode splitMode,
    hipStream_t stream
) {
  constexpr unsigned NUM_WARPS = 10;
  const unsigned numBlocks = std::min(
      std::max(
          ceil_div<unsigned>(numExperts, NUM_WARPS), (unsigned)(maxNumTokens * expertsPerToken)
      ),
      132u
  );
  dim3 dimGrid(numBlocks, 1, 1);
  dim3 dimBlock(NUM_WARPS * 32, 1, 1);

  void *args[] = {
      const_cast<int32_t **>(&outNumTokensPerExpert.data),
      const_cast<size_t *>(&outNumTokensPerExpert.strideElem),
      const_cast<std::byte **>(&expertX.data),
      const_cast<size_t *>(&expertX.strideElem),
      const_cast<size_t *>(&expertX.strideRow),
      const_cast<std::byte **>(&expertXScale.data),
      const_cast<size_t *>(&expertXScale.strideElem),
      const_cast<size_t *>(&expertXScale.strideRow),
      const_cast<std::byte **>(&dpX.data),
      const_cast<size_t *>(&dpX.strideElem),
      const_cast<std::byte **>(&dpXScale.data),
      const_cast<size_t *>(&dpXScale.strideElem),
      const_cast<uint32_t **>(&indices.data),
      const_cast<size_t *>(&indices.strideElem),
      const_cast<size_t *>(&indices.strideRow),
      const_cast<size_t *>(&maxNumTokens),
      const_cast<size_t *>(&numExperts),
      const_cast<unsigned *>(&rank),
      const_cast<unsigned *>(&worldSize),
      const_cast<unsigned *>(&dpSize),
      const_cast<size_t *>(&hiddenDimBytes),
      const_cast<size_t *>(&hiddenDimScaleBytes),
      const_cast<size_t *>(&expertsPerToken),
      const_cast<unsigned **>(&boundM),
      &m,
      &numTokensPerDP,
      &sourceExpert,
      &sourceIndex,
      &sourceOffset,
      &sourceGroup,
      &numTokensBuffer,
      &numDispatchRecvBuffer,
      &xDispatchIn,
      &xDispatchOut,
  };

  nvtxRangePush("dispatch");
  switch (splitMode) {
  case SplitMode::SEND:
    CUDACHECK(hipLaunchKernel(
        (void *)&dispatchKernel<NUM_WARPS, true, false>,
        dimGrid,
        dimBlock,
        args,
        sizeof(uint32_t) * numExperts,
        stream
    ));
    break;
  case SplitMode::RECV:
    CUDACHECK(hipLaunchCooperativeKernel(
        (void *)&dispatchKernel<NUM_WARPS, false, true>, dimGrid, dimBlock, args, 0, stream
    ));
    break;
  case SplitMode::NONE:
    CUDACHECK(hipLaunchCooperativeKernel(
        (void *)&dispatchKernel<NUM_WARPS, true, true>,
        dimGrid,
        dimBlock,
        args,
        sizeof(uint32_t) * numExperts,
        stream
    ));
    break;
  default:
    ROSE_UNREACHABLE("invalid split mode");
  }
  nvtxRangePop();
}

