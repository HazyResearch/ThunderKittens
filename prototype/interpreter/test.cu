#include "interpreter.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::interpreter;
struct config {
    struct globals {
        using instructions_global = kittens::gl<int, 1, -1, -1, 1>;
        instructions_global instructions;
        int dynamic_shared_memory() { return 224000; }
        dim3 grid()  { return dim3(132); }
        dim3 block() { return dim3((
            kittens::prototype::interpreter::NUM_CONSUMER_WARPS+kittens::prototype::interpreter::NUM_PRODUCER_WARPS)
            * kittens::WARP_THREADS
        ); }
    };
};
template<typename _config> struct OpA {
    using config = _config;
    static constexpr int opcode = 1;
    struct layout {
        using globals = config::globals;
        struct input_block { st_bf<64, 64> tile; };
    };
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        args.num_iters = -1;
        if(threadIdx.x == 0) printf("block %d running op A (%d)\n", blockIdx.x, opcode);
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {
            warpgroup::producer_registers();
        }
        __device__ static inline void load(producer_load_args<layout> args) {
            if(laneid() == 0) arrive(args.inputs_arrived);
            warpgroup::sync(warpgroup::groupid());
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            warpgroup::consumer_registers<4>();
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            if(laneid() == 0) arrive(args.inputs_finished);
            warpgroup::sync(warpgroup::groupid());
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            if(laneid() == 0) arrive(args.finish_finished);
        }
    };
};
template<typename _config> struct OpB {
    using config = _config;
    static constexpr int opcode = 2;
    struct layout {
        using globals = config::globals;
        struct input_block { st_bf<64, 64> tile; };
    };
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        args.num_iters = -1;
        if(threadIdx.x == 0) printf("block %d running op B (%d)\n", blockIdx.x, opcode);
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {
            warpgroup::producer_registers();
        }
        __device__ static inline void load(producer_load_args<layout> args) {
            if(laneid() == 0) arrive(args.inputs_arrived);
            warpgroup::sync(warpgroup::groupid());
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            warpgroup::consumer_registers<4>();
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            if(laneid() == 0) arrive(args.inputs_finished);
            warpgroup::sync(warpgroup::groupid());
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            if(laneid() == 0) arrive(args.finish_finished);
        }
    };
};


int main() {
    constexpr int NUM_INSTRUCTIONS = 5;
    int instructions[NUM_INSTRUCTIONS] = {1, 2, 1, 0, 2}; // last 2 should not execute.
    std::vector<int> instructions_vec(132*NUM_INSTRUCTIONS);
    for(int i = 0; i < 132*NUM_INSTRUCTIONS; i++) {
        instructions_vec[i] = instructions[i % NUM_INSTRUCTIONS];
    }
    int *instructions_d;
    hipMalloc(&instructions_d, sizeof(int) * NUM_INSTRUCTIONS*132);
    hipMemcpy(instructions_d, instructions_vec.data(), sizeof(int) * NUM_INSTRUCTIONS*132, hipMemcpyHostToDevice);
    kittens::gl<int, 1, -1, -1, 1> instructions_gl{instructions_d, nullptr, 132, NUM_INSTRUCTIONS, nullptr};
    config::globals G{instructions_gl};
    kittens::prototype::interpreter::run<config, OpA<config>, OpB<config>>(G);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }
    hipDeviceSynchronize();
    err = hipGetLastError(); 
    if (err != hipSuccess) {
        printf("CUDA error after synchronize: %s\n", hipGetErrorString(err));
        return 1;
    }
}