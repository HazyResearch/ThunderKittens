#include "kittens.cuh"
// #define KVM_DEBUG
#include "vm.cuh"
#include <iostream>

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::vm;

using config = default_config;
struct globals {
    using instruction_layout = ::kittens::prototype::vm::instruction_layout<config>;
    using timing_layout = ::kittens::prototype::vm::timing_layout<config>;
    instruction_layout instructions;
    timing_layout timings;
};

template<typename config=config> struct TestOp {
    static constexpr int opcode = 1;
    static __device__ inline int num_pages(const globals &g, state<config> &s) { return 0; }
    static __device__ inline int num_mini_pages(const globals &g, state<config> &s) { return 0; }
    struct launcher {
        static __device__ void run(const globals &g, state<config> &s) {}
    };
    struct storer {
        static __device__ void run(const globals &g, state<config> &s) {}
    };
    struct loader {
        static __device__ void run(const globals &g, state<config> &s) {}
    };
    struct consumer {
        static __device__ void run(const globals &g, state<config> &s) {}
    };
};

int main() {

    vm::print_config<config>();

    // Initialize a vector of one 1 and 31 0's
    int instruction[config::INSTRUCTION_WIDTH] = {1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    // Create a device array for the instruction
    int *d_instruction;
    hipMalloc(&d_instruction, config::INSTRUCTION_WIDTH * sizeof(int));
    hipMemcpy(d_instruction, instruction, config::INSTRUCTION_WIDTH * sizeof(int), hipMemcpyHostToDevice);

    // Create a device array for timing data
    int *d_timing;
    hipMalloc(&d_timing, config::TIMING_WIDTH * sizeof(int));
    hipMemset(d_timing, 0, config::TIMING_WIDTH * sizeof(int));
    
    // Use the device array
    typename globals::instruction_layout instructions{d_instruction, nullptr, 1, 1, nullptr};
    typename globals::timing_layout timings{d_timing, nullptr, 1, 1, nullptr};
    globals g{instructions, timings};
    ::kittens::prototype::vm::kernel<config, globals, TestOp<config>><<<1, config::NUM_THREADS>>>(g);
    
    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    
    // Synchronize device to ensure all operations are complete
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        std::cerr << "CUDA synchronize error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    // Clean up
    hipFree(d_instruction);
    hipFree(d_timing);

    return 0;
}