#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <assert.h>
#include <mma.h>
using namespace nvcuda; 

# include <cuda/pipeline>
# include <hip/hip_cooperative_groups.h>
# include "src/kittens.cuh"
# include "src/common/pyutils/torch_helpers.cuh"

#include <ATen/cuda/HIPContext.h>  // Include necessary for getting the current stream

using namespace kittens;

__device__
void thread_block_load(st_bf_4x4<ducks::st_layout::xor_swizzle> &_dst, const typename st_bf_4x4<ducks::st_layout::xor_swizzle>::dtype *_src, const int nThreads=256) {
    float4* dst = (float4*) _dst.data;
    float4* src = (float4*) _src; 
    using H     = st_bf_4x4<ducks::st_layout::xor_swizzle>;
    using T     = typename H::dtype;

    const int _row_stride  = H::cols; 
    auto bytes_per_row     = H::cols * sizeof(T); // non-padded
    auto f4_stride         = (_row_stride*sizeof(T))/sizeof(float4);
    auto reads_per_row     = bytes_per_row / sizeof(float4);
    auto rows_per_block    = nThreads / reads_per_row; 
    auto row_skipping_only = (nThreads % reads_per_row) == 0; // if we read complete rows.
    auto f4_elements       = (H::num_elements * sizeof(T)) / sizeof(float4);
    
    if( row_skipping_only ) {
        auto col      = threadIdx.x % reads_per_row; // this will be fixed
        auto row_base = threadIdx.x / reads_per_row; 
        auto _stride  = f4_stride*rows_per_block; // we we will just skip!
        __syncthreads();
        auto idx = row_base*f4_stride + col;
        for(auto i = threadIdx.x; i < f4_elements; i+=nThreads, idx += _stride) {
            dst[idx] = src[i];
        }
    } else {
        __syncthreads();
        for(auto i = threadIdx.x; i < f4_elements; i+=nThreads) {
            auto col = i % reads_per_row;
            auto row = i / reads_per_row;
            dst[row*_row_stride + col] = src[i];
        }
    }
}

template<typename op>
__device__ 
void shm_broadcast(float &f, float *shm, const int workers = 4) {
    auto warpid = threadIdx.x / 32;
    auto lane   = threadIdx.x % 32;
    shm[warpid] = f;
    __syncthreads();
    if(warpid == 0) {
        if(lane == 0) {
            for(auto j = 1; j < workers; j++) {f = op::op(f,shm[j]);}
            for(auto j = 0; j < workers; j++) {shm[j] = f;}
        }
        __syncwarp();
    }
    __syncthreads();
    f = shm[warpid];
}

// GEMV
__device__
void gemv(rt_fl_1x4<>::col_vec  &o, rt_fl_1x4<>::row_vec &x, rt_fl_1x4<> &a) { 
    rt_fl_1x4<> t;
    copy(t, a);
    // The accumulator is row x column; row multiply means that each row is multiplied by a column matrix. 
    mul_col(t, a, x); // multiply vv in place with aa: a * v.unsqueeze(1) // row, row, col
    row_sum(o, t, o); // aa.sum(0) sum across all the rows 
}

// GEMV
__device__
void gemv_two(rt_fl_4x1<>::row_vec  &o, rt_fl_4x1<>::col_vec &x, rt_fl_4x1<> &a) { 
    rt_fl_4x1<> t;
    copy(t, a);
    // The accumulator is row x column; row multiply means that each row is multiplied by a column matrix. 
    // mul_row(t, a, x); // SA: uncommenting this line leads to nans in the output
    col_sum(o, t, o); // aa.sum(0) sum across all the rows 
}


static
void __device__
vec_to_rvec(rt_fl_4x1<>::col_vec &dst, const __hip_bfloat16 *src) {
    using T = __hip_bfloat16;
    using U = float;
    auto row = kittens::laneid() / 4;
    __syncwarp();    
    for(auto h = 0; h < dst.outer_dim; h++) {
        dst[h][0].x = base_types::convertor<U, T>::convert(src[h*kittens::TILE_DIM + row]);    
        dst[h][1].x = base_types::convertor<U, T>::convert(src[h*kittens::TILE_DIM + row + 8]); 
    }
}

static void __device__
rvec_to_vec(__hip_bfloat16 *dst, rt_fl_1x4<>::col_vec &src) {
    using U = __hip_bfloat16;
    using T = float;
    auto row = kittens::laneid() / 4;
    __syncwarp();
    if(kittens::laneid() % 4 == 0) { // only the leaders write
        for(auto h = 0; h < src.outer_dim; h++) {
            dst[h*TILE_DIM + row]     = base_types::convertor<U, T>::convert(src[h][0].x);  
            dst[h*TILE_DIM + row + 8] = base_types::convertor<U, T>::convert(src[h][1].x);
        }
    }    
}


template<typename H, typename T>
__global__
void sliding_window_ker_hack(int n, int j, bool just_q, const T* __q, const T* __k, const T* __v, T* __o) {
    
    auto warpid = kittens::warp_id();
    const int d = 64;
    const int window_size = 64;
    const int workers = 4;
    const int threads = workers * kittens::WARP_SIZE;
    auto head_offset  = blockIdx.x * n * d;
    
    const H* _q = device_cast(__q) + blockIdx.x*d;
    const H* _k = device_cast(__k) + head_offset;
    const H* _v = device_cast(__v) + head_offset;
          H* _o = device_cast(__o) + blockIdx.x*d; // just a single vector

    // Register
    rt_fl_1x4<> k_slice; 
    rt_fl_4x1<> v_slice; // Each of the 4 workers stores a column
    rt_fl_1x4<>::row_vec qv; // full local copy 
    rt_fl_1x4<>::col_vec ws; 
    rt_fl_4x1<>::col_vec wv; // full local copy 
    rt_fl_4x1<>::row_vec os; // shards
    auto vec_idx = 0;
    __syncthreads();
    load(qv, _q + vec_idx); // every warp gets a full copy of q; These are column slices of the matrix.: | K_1 | K_2 | K_3 |

    // Shared
    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);
    st_bf_1x4<ducks::st_layout::xor_swizzle>::row_vec &w = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>::row_vec>();
    __shared__ float _max[workers], _sum[workers];  

    // Option A (References / Following the tests)
    const auto start_idx = 0;
    st_bf_4x4<ducks::st_layout::xor_swizzle> &k = al.allocate<st_bf_4x4<ducks::st_layout::xor_swizzle>>(); // We use 4x4 since 4x16 is 64 window size
    st_bf_4x4<ducks::st_layout::xor_swizzle> &v = al.allocate<st_bf_4x4<ducks::st_layout::xor_swizzle>>();
    if(warpid == 0) load(k, _k + start_idx, d); // One warp loads from global to shared
    if(warpid == 0) load(v, _v + start_idx, d);
    __syncthreads();
    auto subtile = k.template subtile<1,4>(warpid, 0); // All the other warps load from shared to shared
    load(k_slice, subtile);

    // Option B
    // st_bf_4x4<ducks::st_layout::xor_swizzle> k = al.allocate<st_bf_4x4<ducks::st_layout::xor_swizzle>>(); // We use 4x4 since 4x16 is 64 window size
    // st_bf_4x4<ducks::st_layout::xor_swizzle> v = al.allocate<st_bf_4x4<ducks::st_layout::xor_swizzle>>();
    // thread_block_load(k, _k + start_idx, threads); 
    // thread_block_load(v, _v + start_idx, threads);   
    // auto subtile = k.template subtile<1,4>(warpid, 0); 
    // load(k_slice, subtile); // SA: Uncommenting this leads to static asserts in the output (even if i uncomment the thread_block_loads)
    __syncthreads();


    one(k_slice);
    one(v_slice);


    // The algorithm.
    // qs = [q for j in range(4)] # broadcast q to each warp
    // ks = [k[:,j*d//4:(j+1)*d//4] for j in range(4)] # shard k
    // ws = [torch.einsum("d, de->e", qs[j],ks[j]) for j in range(4)]
    zero(ws);
    gemv(ws, qv, k_slice);

    // local_max = [ws[j].max() for j in range(4)] # compute local, then global max
    // the_max = torch.tensor(local_max).max()
    float local_max= -INFINITY;
    max(ws, ws, local_max);
    shm_broadcast<base_ops::mul>(local_max, _max);
    
    // ews = [torch.exp(ws[j] - the_max) for j in range(4)]
    sub(ws, ws, local_max);
    exp(ws, ws);

    // es  = [ews[j].sum() for j in range(4)]
    float local_sum = 0.f;
    add(ws, ws, local_sum);
    shm_broadcast<base_ops::sum>(local_sum, _sum);
    
    // w  /= the_sum
    div(ws, ws, local_sum);

    // broadcast w back to shared memory
    rvec_to_vec(&w.data[warpid*kittens::TILE_DIM], ws);
    __syncthreads(); // let the writes complete
    vec_to_rvec(wv, w.data); // read the *whole* v here.
    
    // we want a column stripe of V
    auto subtile_v = v.template subtile<4,1>(0, warpid);
    // load(v_slice, subtile_v); // SA: Uncommenting this leads to static asserts in the output (even if i uncomment the thread_block_loads)
    zero(os);
    gemv_two(os, wv, v_slice);
    
    // now we have a fragment of v and we write, this write is to *global* memory.
    store(_o + warpid*kittens::TILE_DIM, os);
}

void 
sliding_window(int j,   
    torch::Tensor q, torch::Tensor k, torch::Tensor v, 
    torch::Tensor o) {

    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(o);
    
    uint batch = q.size(0);
    uint head  = q.size(1);
    uint d     = q.size(3);
    TORCH_CHECK(d == 64, "Only dimension 64 implemented...");

    bool k_same = true, v_same = true;
    for(auto i = 0; i < 2; i++) { 
        k_same &= q.size(i) == k.size(i);
        v_same &= q.size(i) == v.size(i);
    }
    k_same &= d == k.size(3);
    v_same &= d == v.size(3);
    uint n     = k.size(2);
    v_same &= v.size(2) == n;

    // This is just a restriction of what we're doing now...
    TORCH_CHECK(k_same, "X and K_out should be same size");
    TORCH_CHECK(v_same, "X and V_out should be same size");
    
    const int workers = 4;
    using H = __hip_bfloat16;
    using T = c10::BFloat16;

    auto threads = workers * kittens::WARP_SIZE;

    auto stream_wrapper = at::cuda::getCurrentCUDAStream(q.device().index());
    hipStream_t stream = stream_wrapper.stream();
    sliding_window_ker_hack<H,T><<<batch*head,threads,0,stream>>>(n, j, q.size(2) == 1,
                        q.data_ptr<T>(), k.data_ptr<T>(), v.data_ptr<T>(), 
                        o.data_ptr<T>());
}

