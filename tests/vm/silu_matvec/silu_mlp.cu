#include "hip/hip_runtime.h"
#define RED_TEXT "\033[31m"
#define GREEN_TEXT "\033[32m"
#define YELLOW_TEXT "\033[33m"
#define BLUE_TEXT "\033[34m"
#define MAGENTA_TEXT "\033[35m"
#define CYAN_TEXT "\033[36m"
#define WHITE_TEXT "\033[37m"
#define RESET_TEXT "\033[0m"

#include "kittens.cuh"
// #define KVM_DEBUG
#include "vm/vm.cuh"
#include <iostream>

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::vm;


using config = default_config;
struct globals
{
    using instruction_layout = ::kittens::prototype::vm::instruction_layout<config>;
    using timing_layout = ::kittens::prototype::vm::timing_layout<config>;
    using weights = gl<bf16, 1, -1, -1, 2048, st_bf<16, 512>>; // assumed to be N by 2048 (X@W.T).
    using activations = gl<bf16, 1, 1, 1, 2048, sv_bf<2048>, sv_bf<16>>;
    using barriers = gl<bf16, 1, -1, 6, 32>; // num_layers by 6 ops per layer by up to 32 heads.
    
    instruction_layout instructions;
    timing_layout timings;

    weights UP_PROJ_W;
    weights GATE_PROJ_W;
    activations INP;
    activations O;
    barriers Bar;
    
    // persistent grid structure
    dim3 grid() { return dim3(148); } 
    dim3 block() { return dim3(config::NUM_THREADS); }
    int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }
};

template <typename config = config, int _OP_IDX = 0>
struct SiLU_MLPOp
{
    static constexpr int opcode = 3;
    static constexpr int OP_IDX = _OP_IDX; // Op index within the layer -- controls which barrier to listen to.
    struct parsed_instruction
    {
        int layer, start_col;
        __device__ inline parsed_instruction(typename config::instruction_t &instruction)
        {
            layer = instruction[1];     // in units of 1
            start_col = instruction[2]; // in units of 1
        }
        __device__ inline parsed_instruction(state<config> &s) : parsed_instruction(s.instruction()) {}
    };
    static __device__ inline parsed_instruction parse_instruction(const globals &g, state<config> &s)
    {
        return parsed_instruction{s.instruction()[1], s.instruction()[2]};
    }

    static constexpr int UP_PAGES    = 4;
    static constexpr int GATE_PAGES  = 4;
    static constexpr int PAGE_INPUT  = UP_PAGES + GATE_PAGES;    // = 8
    static constexpr int PAGE_OUTPUT = PAGE_INPUT + 1;           // = 9
    static constexpr int SEM_COUNT   = PAGE_OUTPUT + 1;          // = 10

    //  semaphores 
  __device__ static inline semaphore &up_arrived   (state<config> &s, int i) { return s.semaphores()[ i            ]; }
  __device__ static inline semaphore &gate_arrived (state<config> &s, int i) { return s.semaphores()[ UP_PAGES + i ]; }
  __device__ static inline semaphore &in_arrived   (state<config> &s)        { return s.semaphores()[ PAGE_INPUT   ]; }
  __device__ static inline semaphore &out_arrived  (state<config> &s)        { return s.semaphores()[ PAGE_OUTPUT  ]; }

    // getters
    __device__ static inline int get_up_page  (state<config> &s, int i) { return s.pid(i); }
    __device__ static inline int get_gate_page(state<config> &s, int i) { return s.pid(UP_PAGES + i); }
    __device__ static inline int get_input_page(state<config> &s) { return s.pid(PAGE_INPUT); }
    __device__ static inline int get_output_page(state<config> &s) { return s.pid(PAGE_OUTPUT); }

    struct controller
    {
        static __device__ int release_lid(const globals &g, typename config::instruction_t &instruction, int &query)
        {
            int ret_order[] = {
                6, 7, 8, 9, 10, 11, 12, 
                13,
                0, 1, 2, 3, 4, 5
            };
            
            if ( laneid() == 0 && warpgroup::warpid() == 0 ) { 
                printf("Inside controller release lid!\n");
            }
            return ret_order[query];
        }
        static __device__ int init_semaphores(const globals &g, state<config> &s)
        {

            if ( laneid() == 0 && warpgroup::warpid() == 0 ) { 
                printf("Inside controller init semaphores!\n");
            }

            // each weight page and the input page needs exactly 1 “ready” signal
            for (int i = 0; i < UP_PAGES;   i++) init_semaphore(up_arrived(s,i),   1);
            for (int i = 0; i < GATE_PAGES; i++) init_semaphore(gate_arrived(s,i), 1);
            init_semaphore(in_arrived(s),   1);
            // output must wait for all 4 consumer warps
            init_semaphore(out_arrived(s),  16);
            
            return SEM_COUNT;
        }
    };


    struct loader
    {
        static __device__ void run(const globals &g, state<config> &s)
        {

            if ( laneid() == 0 && warpgroup::warpid() == 0 ) { 
                printf("Inside loader run!\n");
            }

            parsed_instruction inst{s};
            // clear scratch buffer
            ((int*)s.scratch())[laneid()] = 0;
            warp::sync();

            // 1) UP projections
            if (laneid() < UP_PAGES)
            {
                int pg = get_up_page(s, laneid());
                s.wait_page_ready(pg);
                s.record(16 + laneid());
                auto &chunk = reinterpret_cast<st_bf<16,512>&>(s.pages[pg]);
                tma::expect(up_arrived(s,laneid()), chunk);
                tma::load_async(chunk, g.UP_PROJ_W,
                                {inst.layer, inst.start_col/16, laneid()},
                                up_arrived(s,laneid()));
            }

            // 2) GATE projections
            else if (laneid() < UP_PAGES + GATE_PAGES)
            {
                int idx = laneid() - UP_PAGES;
                int pg  = get_gate_page(s, idx);
                s.wait_page_ready(pg);
                s.record(16 + laneid());
                auto &chunk = reinterpret_cast<st_bf<16,512>&>(s.pages[pg]);
                tma::expect(gate_arrived(s,idx), chunk);
                tma::load_async(chunk, g.GATE_PROJ_W,
                                {inst.layer, inst.start_col/16, idx},
                                gate_arrived(s,idx));
            }

            // 4) INPUT page
            else if (laneid() == PAGE_INPUT)
            {
                int pg = get_input_page(s);
                s.wait_page_ready(pg);
                // wait on barrier from previous op
                while (*(volatile int*)&g.Bar[{inst.layer, OP_IDX, 0}] == 0)
                    __nanosleep(20);
                s.record(24);
                auto &buf = reinterpret_cast<sv_bf<2048>&>(s.pages[pg]);
                tma::expect(in_arrived(s), buf);
                tma::load_async(buf, g.INP, {}, in_arrived(s));
            }

            // 5) UNUSED pages: release them immediately so consumer warps can retire
            else if (laneid() >= PAGE_INPUT+1 && laneid() < SEM_COUNT)
            {
                int pg = s.pid(laneid());
                s.wait_page_ready(pg);
                arrive(s.page_finished[pg], config::NUM_CONSUMER_WARPS);
            }
        }
    };


    struct launcher
    { // launches mma's
        // launcher does nothing here, since this doesn't use tensor cores.
        static __device__ void run(const globals &g, state<config> &s)
        {
            // printf("launcher at %d %d\n", laneid(), warpid());
            s.wait_tensor_ready();
            if (laneid() == 0)
                arrive(s.tensor_finished, config::NUM_CONSUMER_WARPS);
        }
    };


    struct consumer
    {
        static __device__ void run(const globals &g, state<config> &s)
        {
            int group  = warpgroup::groupid();     // which weight‐page group
            int warpid = warpgroup::warpid();      // which “lane‐block”
            int lid    = laneid();                 // 0–31

            if ( laneid() == 0 && warpid == 0 ) { 
                printf("Inside consumer run!\n");
            }

            //--------------------------------------------------
            // 1) LOAD INPUT ACTIVATIONS
            //--------------------------------------------------
            wait(in_arrived(s), 0);
            int in_pg = get_input_page(s);
            typename rt_bf<16, 128>::row_vec x_vec;
            // copy the 16×128bfslice out of shared pages
            sv_bf<128>(&in_smem)[16] = reinterpret_cast<sv_bf<128>(&)[16]>(s.pages[in_pg]);
            warp::load(x_vec, in_smem[warp::warpid()]);
            warp::sync();
            warp::arrive(s.page_finished[in_pg]); // just 1 is sufficient


            //--------------------------------------------------
            // 2) UP PROJECTION
            //--------------------------------------------------
            wait(up_arrived(s, group), 0);
            int up_pg = get_up_page(s, group);
            st_bf<16,128> up_smem[4];
            memcpy(up_smem, &s.pages[up_pg], sizeof(up_smem));
            rt_bf<16,128> up_reg;
            warp::load(up_reg, up_smem[warpid]);
            warp::sync();
            warp::arrive(s.page_finished[up_pg], config::NUM_CONSUMER_WARPS);

            // broadcast & mul
            rt_bf<16,128> acc;
            warp::broadcast_col(acc, x_vec);
            warp::mul(acc, acc, up_reg);


            //--------------------------------------------------
            // 3) GATE PROJECTION
            //--------------------------------------------------
            wait(gate_arrived(s, group), 0);
            int gate_pg = get_gate_page(s, group);
            st_bf<16,128> gate_smem[4];
            memcpy(gate_smem, &s.pages[gate_pg], sizeof(gate_smem));
            rt_bf<16,128> gate_reg;
            warp::load(gate_reg, gate_smem[warpid]);
            warp::sync();
            warp::arrive(s.page_finished[gate_pg], config::NUM_CONSUMER_WARPS);

            // mul in place
            warp::mul(acc, acc, gate_reg);


            //--------------------------------------------------
            // 4) SiLU FUSION (in‑place on acc)
            //--------------------------------------------------
            #pragma unroll
            for (int i = 0; i < 16; ++i) {
                #pragma unroll
                for (int j = 0; j < 4; ++j) {
                    auto & d = acc.tiles[0][i].data[j];
                    float  f0 = __bfloat162float(d.x);
                    float  f1 = __bfloat162float(d.y);
                    float  s0 = f0/(1+expf(-f0));
                    float  s1 = f1/(1+expf(-f1));
                    d.x = __float2bfloat16(f0 * s0);
                    d.y = __float2bfloat16(f1 * s1);
                }
            }
            warp::sync();

            //--------------------------------------------------
            // 5) ROW‐SUM → out_vec (16 lanes)
            //--------------------------------------------------
            rt_bf<16,128>::col_vec col;
            rv_bf<16> out_vec;
            warp::row_sum(col, acc);
            warp::copy(out_vec, col);
            warp::sync();

            // --------------------------------------------------
            // 6) ATOMIC ADD EACH LANE INTO SCRATCH
            // --------------------------------------------------
            // Now the first 16 threads have the output.
            if (laneid() < 16)
            { // this might be a bad idea but yolo, it's probably an okay start
                // and fortunately this is code where ncu will tell us if it's bad..
                // atomicAdd(&((bf16 *)s.scratch())[laneid()], out_vec[0][0]);
            }
            warp::sync();
            warp::arrive(out_arrived(s));
        }
    };


    struct storer
    {
        static __device__ void run(const globals &g, state<config> &s) {
            
            if ( laneid() == 0 && warpgroup::warpid() == 0 ) { 
                printf("Inside storer run!\n");
            }

            parsed_instruction inst{s};

            if (laneid() == 0) {
                // wait for all consumer warps
                wait(out_arrived(s), 0);
                // read back the float sums
                void *scratch = s.scratch();
                // now treat that flat array as an sv_bf<16> tile
                sv_bf<16> &output = *reinterpret_cast<sv_bf<16> *>(scratch);
                tma::store_async(g.O, output, { inst.start_col/16 });
                tma::store_async_wait();
            }

            warp::sync();
            asm volatile("fence.acq_rel.gpu;\n");
            if (laneid() == 0) {
                if constexpr (OP_IDX == g.Bar.rows() - 1)
                    atomicAdd(&g.Bar[{inst.layer + 1, 0, 0}], 1);
                else
                    atomicAdd(&g.Bar[{inst.layer, OP_IDX + 1, 0}], 1);
            }
        }
    };
};

#include "pyutils/pyutils.cuh"

PYBIND11_MODULE(silu_mlp, m)
{
    m.doc() = "silu_mlp python module";
    kittens::py::bind_kernel<kvm<config, globals, SiLU_MLPOp<config>>>(
        m, "silu_mlp",
        &globals::instructions,
        &globals::timings,
        &globals::UP_PROJ_W,
        &globals::GATE_PROJ_W,
        &globals::INP,
        &globals::O,
        &globals::Bar
    );
    hipGetLastError();
}
