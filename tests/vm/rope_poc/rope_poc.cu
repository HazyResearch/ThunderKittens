#include "hip/hip_runtime.h"
#include <iostream>

#include "kittens.cuh"
#include "vm/vm.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::vm;

constexpr int TEMP_OPCODE = 1; // this is not full instruction
constexpr int NUM_BLOCKS = 148;
constexpr int QKV_BLOCK_SIZE = 16;
constexpr int HEAD_DIM = 64;
constexpr int NUM_Q_HEADS = 32;
constexpr int NUM_KV_HEADS = 8;

using qkv_rope_rv = rv_bf<16>;
using qkv_rope_sv = sv_bf<16>;

using config = default_config;
struct globals {
    using instruction_layout = ::kittens::prototype::vm::instruction_layout<config>;
    using timing_layout = ::kittens::prototype::vm::timing_layout<config>;
    using qkv_proj_layout = gl<bf16, 1, 1, 1, (NUM_Q_HEADS + NUM_KV_HEADS * 2) * HEAD_DIM, qkv_rope_sv>;
    using rope_layout = gl<bf16, 1, 1, -1, HEAD_DIM, qkv_rope_sv>; // (N_max, D_h)
    using q_layout = gl<bf16, 1, 1, 1, NUM_Q_HEADS * HEAD_DIM, qkv_rope_sv>;
    using kv_layout = gl<bf16, -1, -1, NUM_KV_HEADS, HEAD_DIM, qkv_rope_sv>; // (L, N_max, H_kv, D_h)
    instruction_layout instructions;
    timing_layout timings;
    qkv_proj_layout QKV_proj;
    rope_layout rope_cos;
    rope_layout rope_sin;
    q_layout Q;
    kv_layout K_c;
    kv_layout V_c;
    int pos_id;
    dim3 grid() { return dim3(NUM_BLOCKS); }
    dim3 block() { return dim3(config::NUM_THREADS); }
    int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }
};

template<typename config=config> struct rope_op {
    static constexpr int opcode = TEMP_OPCODE;
    static constexpr int NUM_Q_ELEMS = NUM_Q_HEADS * HEAD_DIM;
    static constexpr int NUM_QK_ELEMS = NUM_Q_ELEMS + NUM_KV_HEADS * HEAD_DIM;
    static constexpr int Q_BLK_START = 0; // I like consistency
    static constexpr int K_BLK_START = NUM_Q_ELEMS / QKV_BLOCK_SIZE;
    static constexpr int V_BLK_START = NUM_QK_ELEMS / QKV_BLOCK_SIZE;

    struct parsed_instruction {
        int layer_idx;
        int qkv_block_idx;
        __device__ inline parsed_instruction(typename config::instruction_t &instruction) {
            layer_idx = instruction[1];
            qkv_block_idx = instruction[2]; // 16 elements per block!
        }
        __device__ inline parsed_instruction(state<config> &s): parsed_instruction(s.instruction()) {}
    };

    // Semaphores
    __device__ static inline semaphore &setup_ready(state<config> &s) { return s.semaphores()[0]; }
    __device__ static inline semaphore &rope_cos_ready(state<config> &s) { return s.semaphores()[1]; }
    __device__ static inline semaphore &rope_sin_ready(state<config> &s) { return s.semaphores()[2]; }
    __device__ static inline semaphore &output_ready(state<config> &s) { return s.semaphores()[3]; }

    // Pages (very naive for now, no fine-grained usage)
    __device__ static inline int get_QKV_proj_page(state<config> &s) { return s.pid(0); }
    __device__ static inline int get_rope_cos_page(state<config> &s) { return s.pid(1); }
    __device__ static inline int get_rope_sin_page(state<config> &s) { return s.pid(2); }

    struct controller {
        static __device__ int release_lid(const globals &g, typename config::instruction_t &instruction, int &query) {
            int ret_order[13] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
            return ret_order[query];
        }
        static __device__ int init_semaphores(const globals &g, state<config> &s) {
            init_semaphore(setup_ready(s), 0, 1);
            init_semaphore(rope_cos_ready(s), 0, 1);
            init_semaphore(rope_sin_ready(s), 0, 1);
            init_semaphore(output_ready(s), 0, 1);
            return 4;
        }
    };
    struct loader {
        static __device__ void run(const globals &g, state<config> &s) {
            parsed_instruction inst{s};
            if (warp::laneid() >= 6 && warp::laneid() < 13)
                arrive(s.page_finished[s.pid(warp::laneid())], config::NUM_CONSUMER_WARPS); // Release the unused page immediately



            /* THIS PART DOESN'T EXIST IN FUSED OP */
            /* THIS PART DOESN'T EXIST IN FUSED OP */
            if (warp::laneid() == 0) {
                int qkv_proj_page_idx = get_QKV_proj_page(s);
                s.wait_page_ready(qkv_proj_page_idx);
                qkv_rope_sv &qkv_proj_smem = reinterpret_cast<qkv_rope_sv &>(s.pages[qkv_proj_page_idx]);
                tma::expect(setup_ready(s), qkv_proj_smem);
                tma::load_async(qkv_proj_smem, g.QKV_proj, {0, 0, 0, inst.qkv_block_idx}, setup_ready(s));
                wait(setup_ready(s), 0);
            }
            warp::sync();
            /* THIS PART DOESN'T EXIST IN FUSED OP */
            /* THIS PART DOESN'T EXIST IN FUSED OP */



            // Load rotary encodings
            int rope_dim_idx = inst.qkv_block_idx % 4; // 0, 1, 2, 3
            if (warp::laneid() == 0) {
                int rope_cos_page_idx = get_rope_cos_page(s);
                s.wait_page_ready(rope_cos_page_idx);
                qkv_rope_sv &rope_cos_smem = reinterpret_cast<qkv_rope_sv &>(s.pages[rope_cos_page_idx]);
                tma::expect(rope_cos_ready(s), rope_cos_smem);
                tma::load_async(rope_cos_smem, g.rope_cos, {0, 0, g.pos_id, rope_dim_idx}, rope_cos_ready(s));
            } else if (warp::laneid() == 1) {
                int rope_sin_page_idx = get_rope_sin_page(s);
                s.wait_page_ready(rope_sin_page_idx);
                qkv_rope_sv &rope_sin_smem = reinterpret_cast<qkv_rope_sv &>(s.pages[rope_sin_page_idx]);
                tma::expect(rope_sin_ready(s), rope_sin_smem);
                tma::load_async(rope_sin_smem, g.rope_sin, {0, 0, g.pos_id, rope_dim_idx}, rope_sin_ready(s));
            }
        }
    };
    struct launcher {
        static __device__ void run(const globals &g, state<config> &s) { }
    };
    struct consumer {
        static __device__ void run(const globals &g, state<config> &s) {
            // Use a single warp
            if (warpid() == 0) {
                parsed_instruction inst{s};
    
                if (inst.qkv_block_idx < V_BLK_START) { // Q or K
                    int qkv_proj_page_idx = get_QKV_proj_page(s);
                    int rope_cos_page_idx = get_rope_cos_page(s);
                    int rope_sin_page_idx = get_rope_sin_page(s);
                    qkv_rope_sv &qkv_proj_smem = reinterpret_cast<qkv_rope_sv &>(s.pages[qkv_proj_page_idx]);
                    qkv_rope_sv &rope_cos_smem = reinterpret_cast<qkv_rope_sv &>(s.pages[rope_cos_page_idx]);
                    qkv_rope_sv &rope_sin_smem = reinterpret_cast<qkv_rope_sv &>(s.pages[rope_sin_page_idx]);
                    qkv_rope_rv qkv_proj_reg;
                    qkv_rope_rv rope_cos_reg;
                    qkv_rope_rv rope_sin_reg;

                    // Load values
                    wait(rope_cos_ready(s), 0);
                    warp::load(qkv_proj_reg, qkv_proj_smem); // for this implementation only, we rely on rope_cos_ready (no need to wait in the fused op)
                    warp::load(rope_cos_reg, rope_cos_smem);
                    wait(rope_sin_ready(s), 0);
                    warp::load(rope_sin_reg, rope_sin_smem);

                    // Release pages
                    warp::arrive(s.page_finished[rope_cos_page_idx], config::NUM_CONSUMER_WARPS);
                    warp::arrive(s.page_finished[rope_sin_page_idx], config::NUM_CONSUMER_WARPS);
                    
                    // Fetch the neighbor values
                    int mod = (laneid() & 0b1) ? -1 : 1; // 1 for even, -1 for odd
                    bf16 pair_val = __shfl_sync(MASK_ALL, qkv_proj_reg[0][0], laneid() + mod);
    
                    // Compute RoPE in-place
                    if (laneid() < 16)
                        qkv_proj_reg[0][0] = qkv_proj_reg[0][0] * rope_cos_reg[0][0] + bf16(-1 * mod) * pair_val * rope_sin_reg[0][0];
    
                    // Store the result in-place
                    warp::store(qkv_proj_smem, qkv_proj_reg);
                    warp::sync();
                } else { // V
                    wait(rope_cos_ready(s), 0); // delete this in fused op (no need to wait for projection, as it is already in reg/smem)
                }

                warp::arrive(output_ready(s));
            }
        }
    };
    struct storer {
        static __device__ void run(const globals &g, state<config> &s) {
            if (warp::laneid() == 0) {
                parsed_instruction inst{s};

                int qkv_proj_page_idx = get_QKV_proj_page(s);
                qkv_rope_sv &qkv_proj_smem = reinterpret_cast<qkv_rope_sv &>(s.pages[qkv_proj_page_idx]);
                wait(output_ready(s), 0);
    
                if (inst.qkv_block_idx < K_BLK_START) { // Q
                    tma::store_async<cache_policy::NORMAL>(g.Q, qkv_proj_smem, {0, 0, 0, inst.qkv_block_idx});
                } else if (inst.qkv_block_idx < V_BLK_START) { // K
                    int base_index = (inst.qkv_block_idx - K_BLK_START) * QKV_BLOCK_SIZE;
                    int head_idx = base_index / HEAD_DIM;
                    int dim_idx = (base_index % HEAD_DIM) / QKV_BLOCK_SIZE;
                    tma::store_async<cache_policy::NORMAL>(g.K_c, qkv_proj_smem, {inst.layer_idx, g.pos_id, head_idx, dim_idx});
                } else { // V
                    int base_index = (inst.qkv_block_idx - V_BLK_START) * QKV_BLOCK_SIZE;
                    int head_idx = base_index / HEAD_DIM;
                    int dim_idx = (base_index % HEAD_DIM) / QKV_BLOCK_SIZE;
                    tma::store_async<cache_policy::NORMAL>(g.V_c, qkv_proj_smem, {inst.layer_idx, g.pos_id, head_idx, dim_idx});
                }

                tma::store_async_read_wait();
                arrive(s.page_finished[qkv_proj_page_idx], config::NUM_CONSUMER_WARPS);
            }
        }
    };
};

#include "pyutils/pyutils.cuh"

PYBIND11_MODULE(rope_poc, m) {
    m.doc() = "";
    kittens::py::bind_kernel<kvm<config, globals, rope_op<config>>>(m, "rope_poc",
        &globals::instructions,
        &globals::timings,
        &globals::QKV_proj,
        &globals::rope_cos,
        &globals::rope_sin,
        &globals::Q,
        &globals::K_c,
        &globals::V_c,
        &globals::pos_id
    );
}
