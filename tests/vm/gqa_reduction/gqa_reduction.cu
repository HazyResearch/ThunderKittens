#include "hip/hip_runtime.h"
#include <iostream>

#include "kittens.cuh"
#include "vm/vm.cuh" // Use the Kittens Virtual Machine framework
#include <limits>

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::vm;

// --- Constants ---
constexpr int GQA_REDUCTION_OPCODE = 2;
constexpr int HEAD_DIM = 64;
constexpr int NUM_Q_HEADS = 32;

using l_partial_sv = sv_fl<16>; //  (only index [0] is relevant)

using o_sv = sv_fl<HEAD_DIM>;
using o_rv = rv_fl<HEAD_DIM>;
using o_vector_rt = rt_fl<16, HEAD_DIM>;
using o_partial_st = st_fl<16, HEAD_DIM>; // Store O partials (only row [0] is relevant)
using o_final_st = st_bf<16, HEAD_DIM>; // Store final O output (only row [0] is relevant)
using o_final_sv = sv_bf<HEAD_DIM>;

using config = default_config;
struct globals {
    using instruction_layout = ::kittens::prototype::vm::instruction_layout<config>;
    using timing_layout = ::kittens::prototype::vm::timing_layout<config>;
    // Input Partial LSE:
    using l_partial_layout = gl<float, 1, 1, NUM_Q_HEADS, -1, l_partial_sv>;
    // Input Partial O:
    using o_partial_layout = gl<float, 1, NUM_Q_HEADS, -1, HEAD_DIM, o_sv>;
    // Final Output O:
    using o_final_layout = gl<bf16, 1, NUM_Q_HEADS, 1, HEAD_DIM, o_final_sv>;

    instruction_layout instructions;
    timing_layout timings;
    l_partial_layout L_partials;     // Input: Global partial LSE tensor
    o_partial_layout O_partials;     // Input: Global partial O tensor
    o_final_layout O_final;          // Output: Global final O tensor

    dim3 grid() { return dim3(NUM_Q_HEADS); } // One block per Q head
    dim3 block() { return dim3(config::NUM_THREADS); }
    int dynamic_shared_memory() { return config::DYNAMIC_SHARED_MEMORY; }
};

template<typename config=config> struct rope_gqa_reduction_op {
    static constexpr int opcode = GQA_REDUCTION_OPCODE;
    static constexpr int NUM_STAGES = 4;

    // --- Instruction Parsing  ---
    // [0] = opcode (2)
    // [1] = num_partials
    struct parsed_instruction {
        int num_partials;
        int q_head_idx; // Implicitly blockIdx.x
        __device__ inline parsed_instruction(state<config> &s) {
            num_partials = s.instruction()[1];
            q_head_idx = blockIdx.x;
        }
    };

    // --- Semaphore Access Helpers ---
    __device__ static inline semaphore &L_partial_arrived(state<config> &s, int stage) { return s.semaphores()[stage * 2]; }
    __device__ static inline semaphore &O_partial_arrived(state<config> &s, int stage) { return s.semaphores()[stage * 2 + 1]; }
    __device__ static inline semaphore &L_partial_finished(state<config> &s, int stage) { return s.semaphores()[NUM_STAGES * 2 + stage * 2]; }
    __device__ static inline semaphore &O_partial_finished(state<config> &s, int stage) { return s.semaphores()[NUM_STAGES * 2 + stage * 2 + 1]; }
    __device__ static inline semaphore &final_O_ready(state<config> &s) { return s.semaphores()[NUM_STAGES * 4]; }

    // --- Shared Memory Page Management Helpers ---
    static constexpr int PARTIALS_PAGE = 0; // Page for pipelined partial O/L
    static constexpr int FINAL_O_PAGE = 1;  // Page for final O storage
    __device__ static inline void wait_partials_page(state<config> &s) { s.wait_page_ready(s.pid(PARTIALS_PAGE)); }
    __device__ static inline void wait_final_o_page(state<config> &s) { s.wait_page_ready(s.pid(FINAL_O_PAGE)); }
    __device__ static inline void finish_partials_page(state<config> &s) {
        if (warp::laneid() == 0) arrive(s.page_finished[s.pid(PARTIALS_PAGE)], config::NUM_CONSUMER_WARPS);
    }
    __device__ static inline void finish_final_o_page(state<config> &s) {
        if (warp::laneid() == 0) arrive(s.page_finished[s.pid(FINAL_O_PAGE)], config::NUM_CONSUMER_WARPS);
    }

    // --- Shared Memory Access Helpers ---
    __device__ static inline l_partial_sv &get_L_partial_smem(state<config> &s, int stage) {
        int pid = s.pid(PARTIALS_PAGE);
        char *base_ptr = reinterpret_cast<char *>(s.pages[pid].data);
        size_t offset = stage * (sizeof(l_partial_sv) + sizeof(o_sv));
        return *reinterpret_cast<l_partial_sv*>(base_ptr + offset);
    }
     __device__ static inline o_sv &get_O_partial_smem(state<config> &s, int stage) {
        int pid = s.pid(PARTIALS_PAGE);
        char *base_ptr = reinterpret_cast<char *>(s.pages[pid].data);
        size_t offset = stage * (sizeof(l_partial_sv) + sizeof(o_sv)) + sizeof(l_partial_sv);
        return *reinterpret_cast<o_sv*>(base_ptr + offset);
    }
    __device__ static inline o_final_sv &get_O_final_smem(state<config> &s) {
        int pid = s.pid(FINAL_O_PAGE);
        return *reinterpret_cast<o_final_sv*>(s.pages[pid].data);
    }

    struct controller {
        static __device__ int release_lid(const globals &g, typename config::instruction_t &instruction, int &query) {
            return query;
        }
        static __device__ int init_semaphores(const globals &g, state<config> &s) {
            parsed_instruction inst{s};
            for (int i = 0; i < NUM_STAGES; i++) {
                init_semaphore(L_partial_arrived(s, i), 0, 1);
                init_semaphore(O_partial_arrived(s, i), 0, 1);
                init_semaphore(L_partial_finished(s, i), 0, 1);
                init_semaphore(O_partial_finished(s, i), 0, 1);
            }
            init_semaphore(final_O_ready(s), 0, 1);
            return 4 * NUM_STAGES + 1;
        }
    };

    struct loader {
        static __device__ void run(const globals &g, state<config> &s) {
            parsed_instruction inst{s};
            int laneid = warp::laneid();
            
            if (laneid >= 2 && laneid < config::NUM_PAGES) arrive(s.page_finished[s.pid(laneid)], config::NUM_CONSUMER_WARPS);
            if (laneid == 0) {
                wait_partials_page(s);
                wait_final_o_page(s);

                for (int i = 0; i < inst.num_partials; ++i) {
                    int stage = i % NUM_STAGES;
                    l_partial_sv &L_smem = get_L_partial_smem(s, stage);
                    o_sv &O_smem = get_O_partial_smem(s, stage);

                    if (i >= NUM_STAGES) {
                        wait(L_partial_finished(s, stage), (i / NUM_STAGES - 1) % 2);
                        wait(O_partial_finished(s, stage), (i / NUM_STAGES - 1) % 2);
                    }

                    // Load L_partial[q_head_idx, i]
                    L_smem.data[0] = g.L_partials.raw_ptr[(inst.q_head_idx * g.L_partials.cols()) + i];
                    for (int i = 1; i < 16; ++i) {
                        L_smem.data[i] = 0;
                    }
                    // tma::expect(L_partial_arrived(s, stage), L_smem);
                    // tma::load_async<cache_policy::EVICT_FIRST>(L_smem, g.L_partials, {0, 0, inst.q_head_idx, i}, L_partial_arrived(s, stage));

                    // Load O_partial[q_head_idx, i] into row 0 of the SMEM tile
                    tma::expect(O_partial_arrived(s, stage), O_smem);
                    tma::load_async<cache_policy::EVICT_FIRST>(O_smem, g.O_partials, {0, inst.q_head_idx, i, 0}, O_partial_arrived(s, stage));
                }
            }
            warp::sync();
        }
    };

    struct launcher {
        static __device__ void run(const globals &g, state<config> &s) { }
    };

    struct consumer {
        static __device__ void run(const globals &g, state<config> &s) {
            parsed_instruction inst{s};
            if (warpid() == 0) {
                o_rv O_final_reg;        // Accumulator O vector
                o_rv O_partial_reg;      // Loaded partial O vector

                float L_final_reg;          // Accumulator LSE (log-sum-exp)
                float L_partial_reg;        // Loaded partial LSE
                float L_max_reg;            // Max in the current step
                float L_max_accum_reg;      // Overall max LSE accumulator
                float scale_final_reg;
                float scale_partial_reg;

                warp::zero(O_final_reg);
                L_final_reg = -INFINITY;
                L_max_accum_reg = -INFINITY;

                // --- Reduction Pipeline ---
                for (int i = 0; i < inst.num_partials; ++i) {
                    int stage = i % NUM_STAGES;
                    warp::wait(O_partial_arrived(s, stage), (i / NUM_STAGES) % 2);

                    l_partial_sv &L_smem = get_L_partial_smem(s, stage);
                    o_sv &O_smem = get_O_partial_smem(s, stage);

                    // Load L_partial_reg
                    uint32_t src_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(&L_smem.data[0]));
                    move<float>::lds(L_partial_reg, src_ptr);
                    // Load O_partial_reg
                    warp::load(O_partial_reg, O_smem);
                    
                    // Update L_max_reg accumulator
                    L_max_reg = max(L_final_reg, L_partial_reg);

                    // Calculate scales based on L_max_reg
                    scale_final_reg = exp2f(L_final_reg - L_max_reg);
                    scale_partial_reg = exp2f(L_partial_reg - L_max_reg);

                    // Update O accumulator
                    warp::mul(O_final_reg, O_final_reg, scale_final_reg);
                    warp::mul(O_partial_reg, O_partial_reg, scale_partial_reg);
                    warp::add(O_final_reg, O_final_reg, O_partial_reg);

                    // Update LSE accumulator
                    float sum_scales = scale_final_reg + scale_partial_reg;
                    L_final_reg = L_max_reg + log2f(sum_scales);

                    warp::arrive(L_partial_finished(s, stage));
                    warp::arrive(O_partial_finished(s, stage));
                }
                finish_partials_page(s);

                warp::div(O_final_reg, O_final_reg, exp2f(L_final_reg - L_max_reg));

                o_final_sv &O_final_smem = get_O_final_smem(s);
                warp::store(O_final_smem, O_final_reg);
                warp::sync();

                warp::arrive(final_O_ready(s));
            }
        }
    };

    struct storer {
        static __device__ void run(const globals &g, state<config> &s) {
            parsed_instruction inst{s};
            
            if (warp::laneid() == 0) {
                o_final_sv &O_final_smem = get_O_final_smem(s);
                wait(final_O_ready(s), 0);
    
                tma::store_async<cache_policy::NORMAL>(g.O_final, O_final_smem, {0, inst.q_head_idx, 0, 0});
                tma::store_async_read_wait();
                finish_final_o_page(s);
            }
            warp::sync();
         }
    };
};

#include "pyutils/pyutils.cuh"

// --- Python Bindings ---
PYBIND11_MODULE(gqa_reduction, m) {
    m.doc() = "GQA Reduction VM Operation (Compliant Types)";
    kittens::py::bind_kernel<kvm<config, globals, rope_gqa_reduction_op<config>>>(m, "gqa_reduction",
        &globals::instructions,
        &globals::timings,
        &globals::L_partials,
        &globals::O_partials,
        &globals::O_final
    );
}