#include "kittens.cuh"
// #define KVM_DEBUG
#include "vm/vm.cuh"
#include <iostream>

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::vm;

/*
Instruction format:
[0] = opcode
[1] = Row offset of C, in units of 128
[2] = Col offset of C, in units of 128
[3] = K reduction dimension, in units of 128
*/

using config = default_config;
struct globals {
    using instruction_layout = ::kittens::prototype::vm::instruction_layout<config>;
    using timing_layout = ::kittens::prototype::vm::timing_layout<config>;
    using fp8_matrix = gl<fp8e4m3, 1, 1, -1, -1, st_fl8_e4m3<128, 128>>;
    instruction_layout instructions;
    timing_layout timings;
    fp8_matrix A, B, C;
};

template<typename config=config> struct TestOp {
    static constexpr int opcode = 1;
    struct parsed_instruction {
        int row, col, iters;
    };
    static __device__ inline parsed_instruction parse_instruction(const globals &g, state<config> &s) {
        return parsed_instruction{s.instruction()[1], s.instruction()[2], s.instruction()[3]};
    }
    struct launcher {
        // Uses one minipage, and 4*iters full pages.
        static __device__ void run(const globals &g, state<config> &s) {
            int semaphore_page = s.get_mini_page();
            semaphore *mma_sems = reinterpret_cast<semaphore*>(&s.mini_pages[semaphore_page]);
            init_semaphore(mma_sems[laneid()], 1); // create 32 semaphores for tracking mma lanes, fully cycling every 8 iters.
            parsed_instruction inst = parse_instruction(g, s);
            auto accumulator = s.tensor_alloc.allocate<tt<float, 128, 128>>(laneid()*128);
            int base_mma_lane = laneid() < 4 ? laneid() : laneid()%4;
            if(laneid() < 8) for(int i = 0; i < inst.iters; i++) {
                int a_page, b_page;
                #pragma unroll
                for(int j = 0; j < 2; j++) {
                    int p = s.get_page();
                    if(base_mma_lane/2 == j) a_page = p;
                }
                #pragma unroll
                for(int j = 0; j < 2; j++) {
                    int p = s.get_page();
                    if(base_mma_lane%2 == j) b_page = p;
                }
                int active_mma_lane = (base_mma_lane + 4*i)%32;
                if(laneid() < 4) {
                    wait(mma_sems[active_mma_lane], 1);
                    s.wait_page_arrived(a_page);
                    s.wait_page_arrived(b_page);
                    st_fl8_e4m3<128, 128> &a = s.pages[a_page].as_st<fp8e4m3>();
                    st_fl8_e4m3<128, 128> &b = s.pages[b_page].as_st<fp8e4m3>();
                    mma_AB(accumulator, a, b, mma_sems[active_mma_lane]);
                }
                else if(laneid() < 8) {
                    wait(mma_sems[active_mma_lane], 0);
                    arrive(mma_sems[active_mma_lane]);
                    // Arrive on the relevant page barriers
                    arrive(s.page_finished[a_page], config::NUM_CONSUMER_WARPS/2);
                    arrive(s.page_finished[b_page], config::NUM_CONSUMER_WARPS/2);
                }
            }
            else s.advance_page(4*inst.iters);
            __syncwarp();
            invalidate_semaphore(mma_sems[laneid()]); // Clean up minipage
            // Mark minipage as both arrived and finished.
            warp::arrive(s.mini_page_arrived[semaphore_page], config::NUM_CONSUMER_WARPS);

            s.advance_page(4); // advance 4 pages for the stores.
        }
    };
    struct storer {
        // Uses 4 full pages for outputs.
        static __device__ void run(const globals &g, state<config> &s) {
            parsed_instruction inst = parse_instruction(g, s);
            s.advance_mini_page(1);
            s.advance_page(inst.iters*4);
            for(int r = 0; r < 2; r++) {
                for(int c = 0; c < 2; c++) {
                    int output_page = s.get_page();
                    st_fl8_e4m3<128, 128> &output = s.pages[output_page].as_st<fp8e4m3>();
                    if(laneid() == 2*r+c) {
                        s.wait_page_arrived(output_page);
                        tma::store_async(output, g.C, {inst.row+r, inst.col+c});
                        tma::store_async_read_wait();
                        arrive(s.page_finished[output_page], config::NUM_CONSUMER_WARPS);
                    }
                }
            }
                
        }
    };
    struct loader {
        static __device__ void run(const globals &g, state<config> &s) {
            parsed_instruction inst = parse_instruction(g, s);
            s.advance_mini_page(1);
            for(int i = 0; i < inst.iters; i++) {
                #pragma unroll
                for(int j = 0; j < 2; j++) {
                    int a_page = s.get_page();
                    st_fl8_e4m3<128, 128> &a = s.pages[a_page].as_st<fp8e4m3>();
                    warp::tma::load_async(a, g.A, {inst.row+j, i}, s.page_arrived[a_page]);
                    warp::arrive(s.page_arrived[a_page], config::NUM_CONSUMER_WARPS-1);
                }
                #pragma unroll
                for(int j = 0; j < 2; j++) {
                    int b_page = s.get_page();
                    st_fl8_e4m3<128, 128> &b = s.pages[b_page].as_st<fp8e4m3>();
                    warp::tma::load_async(b, g.B, {i, inst.col+j}, s.page_arrived[b_page]);
                    warp::arrive(s.page_arrived[b_page], config::NUM_CONSUMER_WARPS-1);
                }
            }
            s.advance_page(4); // Advance 4 pages for the stores.
        }
    };
    struct consumer {
        static __device__ void run(const globals &g, state<config> &s) {
            int minipage = s.get_mini_page();
            s.advance_page(inst.iters*4);
            int cons_id = warpgroup::groupid();
            int store_page;
            for(int i = 0; i < 4; i++) {
                int p = s.get_page();
                if(cons_id == i) store_page = p;
            }
            st_fl8_e4m3<128, 128> &store_buffer = s.pages[store_page].as_st<fp8e4m3>();
            s.wait_mini_page_arrived(minipage);
            __syncwarp();
            warp::arrive(s.mini_page_finished[minipage]);
            // Great, now we can start the store.
            auto accumulator = s.tensor_alloc.allocate<tt<float, 128, 128>>(cons_id*128);
            rt_fl<32, 128> acc_rt;
            warpgroup::load_async(acc_rt, accumulator);
            warpgroup::store_async(store_buffer, acc_rt);
            __syncwarp();
            warp::arrive(s.page_arrived[store_page]);
        }
    };
};

int main() {

    vm::print_config<config>();

    // Initialize a vector of a basic instruction.
    int instruction[config::INSTRUCTION_WIDTH] = {1, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    // Create a device array for the instruction
    int *d_instruction;
    hipMalloc(&d_instruction, config::INSTRUCTION_WIDTH * sizeof(int));
    hipMemcpy(d_instruction, instruction, config::INSTRUCTION_WIDTH * sizeof(int), hipMemcpyHostToDevice);

    // Create a device array for timing data
    int *d_timing;
    hipMalloc(&d_timing, config::TIMING_WIDTH * sizeof(int));
    hipMemset(d_timing, 0, config::TIMING_WIDTH * sizeof(int));

    fp8e4m3 *d_A, *d_B, *d_C;
    hipMalloc(&d_A, 128*128*sizeof(fp8e4m3));
    hipMalloc(&d_B, 128*128*sizeof(fp8e4m3));
    hipMalloc(&d_C, 128*128*sizeof(fp8e4m3));
    
    // Use the device array
    typename globals::instruction_layout instructions{d_instruction, nullptr, 1, 1, nullptr};
    typename globals::timing_layout timings{d_timing, nullptr, 1, 1, nullptr};
    globals g{instructions, timings, d_A, d_B, d_C};
    ::kittens::prototype::vm::kernel<config, globals, TestOp<config>><<<1, config::NUM_THREADS>>>(g);
    
    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    
    // Synchronize device to ensure all operations are complete
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        std::cerr << "CUDA synchronize error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }

    // Copy timing data back to host
    int h_timing[config::TIMING_WIDTH];
    hipMemcpy(h_timing, d_timing, config::TIMING_WIDTH * sizeof(int), hipMemcpyDeviceToHost);
    
    // Print all timing data
    std::cout << "Timing data:" << std::endl;
    for (int i = 0; i < config::TIMING_WIDTH; i++) {
        std::cout << "timing[" << i << "] = " << h_timing[i] << std::endl;
    }

    // Clean up
    hipFree(d_instruction);
    hipFree(d_timing);

    std::cout << "Test passed!" << std::endl;

    return 0;
}