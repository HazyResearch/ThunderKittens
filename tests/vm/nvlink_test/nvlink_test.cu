#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"


// CUDA driver API
#define CUCHECK(cmd) do {                                     \
    hipError_t err = cmd;                                       \
    if (err != hipSuccess) {                                \
        const char *errStr;                                   \
        hipDrvGetErrorString(err, &errStr);                       \
        fprintf(stderr, "Failed: CUDA error %s:%d '%s'\n",    \
            __FILE__, __LINE__, errStr);                      \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(0)

// CUDA runtime API
#define CUDACHECK(cmd) do {                                   \
    hipError_t err = cmd;                                    \
    if (err != hipSuccess) {                                 \
        fprintf(stderr, "Failed: CUDA error %s:%d '%s'\n",    \
            __FILE__, __LINE__, hipGetErrorString(err));     \
        exit(EXIT_FAILURE);                                   \
    }                                                         \
} while(0)

using namespace std;

constexpr int STRIDES = 32;
constexpr size_t STRIDE_SIZE = 10 * 1024 * 1024; // 10 MB
constexpr size_t GB = 1024 * 1024 * 1024;
__global__ void kernel(int* src, int* dst, int start) {
    int val = 0;
    if (threadIdx.x == 0) {
        #pragma unroll
        for (size_t i = 0; i < STRIDES; ++i) {
            volatile int _val;
            asm volatile (
                "{ ld.global.u32 %0, [%1]; }"
                : "=r"(_val)
                : "l"(&src[i + start * STRIDE_SIZE * STRIDES / 4])
                : "memory"
            );
            val += _val;
        }
        *dst = val;
    }
    __syncthreads();
}

void benchmark(int *src[2], int *dst[2], int src_dev, int dst_dev, int start, int warmup) {
    hipEvent_t startEvent, stopEvent;
    CUDACHECK(hipSetDevice(dst_dev));
    CUDACHECK(hipEventCreate(&startEvent));
    CUDACHECK(hipEventCreate(&stopEvent));
    CUDACHECK(hipEventRecord(startEvent, 0));
    kernel<<<1, 1, 0, 0>>>(src[src_dev], dst[dst_dev], start); // Can't iter since this tests caching
    CUDACHECK(hipEventRecord(stopEvent, 0));
    CUDACHECK(hipEventSynchronize(stopEvent));
    float elapsedTimeMs;
    CUDACHECK(hipEventElapsedTime(&elapsedTimeMs, startEvent, stopEvent));
    if (!warmup)
        cout << "D" << src_dev << " -> D" << dst_dev << ": " << 1e3 * elapsedTimeMs << " us" << endl;
}

int main() {
    // P2P Setup
    int can_access_peer_0_1;
    int can_access_peer_1_0;
    CUDACHECK(hipDeviceCanAccessPeer(&can_access_peer_0_1, 0, 1));
    CUDACHECK(hipDeviceCanAccessPeer(&can_access_peer_1_0, 1, 0));
    cout << "Device 0 can access device 1: " << can_access_peer_0_1 << endl;
    cout << "Device 1 can access device 0: " << can_access_peer_1_0 << endl;
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipDeviceEnablePeerAccess(1, 0));
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipDeviceEnablePeerAccess(0, 0));

    // Allocate device memory (L2 cache on B200 is 126 MB)
    constexpr size_t SIZE = 2LL * GB;
    int *src[2];
    int *dst[2];
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipMalloc((void**)&src[0], SIZE));
    CUDACHECK(hipMalloc((void**)&dst[0], sizeof(int)));
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMalloc((void**)&src[1], SIZE));
    CUDACHECK(hipMalloc((void**)&dst[1], sizeof(int)));

    // Initialize to random values
    int *h_src[2];
    h_src[0] = new int[SIZE / sizeof(int)];
    h_src[1] = new int[SIZE / sizeof(int)];
    for (size_t i = 0; i < SIZE / sizeof(int); ++i) {
        h_src[0][i] = rand() % 100;
        h_src[1][i] = rand() % 100;
    }
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipMemcpy(src[0], h_src[0], SIZE, hipMemcpyHostToDevice));
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipMemcpy(src[1], h_src[1], SIZE, hipMemcpyHostToDevice));
    delete[] h_src[0];
    delete[] h_src[1];
    
    // Launch benchmarks
    // void benchmark(int *src[2], int *dst[2], int src_dev, int dst_dev, int start, int warmup)
    benchmark(src, dst, 0, 0, 0, 1); // warmup
    benchmark(src, dst, 1, 1, 0, 1); // warmup
    benchmark(src, dst, 1, 0, 0, 1); // warmup
    benchmark(src, dst, 0, 1, 0, 1); // warmup
    benchmark(src, dst, 0, 0, 2, 0); // 0 -> 0
    benchmark(src, dst, 0, 0, 2, 0); // 0 -> 0
    benchmark(src, dst, 1, 0, 4, 0); // 1 -> 0
    benchmark(src, dst, 1, 0, 4, 0); // 1 -> 0
    benchmark(src, dst, 1, 1, 4, 0); // 1 -> 1
    benchmark(src, dst, 1, 1, 4, 0); // 1 -> 1

    // Cleanup
    CUDACHECK(hipSetDevice(0));
    CUDACHECK(hipFree(src[0]));
    CUDACHECK(hipFree(dst[0]));
    CUDACHECK(hipSetDevice(1));
    CUDACHECK(hipFree(src[1]));
    CUDACHECK(hipFree(dst[1]));

    return 0;
}
