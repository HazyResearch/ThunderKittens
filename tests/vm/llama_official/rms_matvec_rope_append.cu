#include "hip/hip_runtime.h"
#include "llama.cuh"
#include "utils.cuh"
#include "matvec_pipeline.cuh"

using namespace kittens;
using namespace kittens::prototype;

namespace kittens::prototype::vm
{

    using globals = llama_1b_globals;

    template <typename Config, typename Globals>
    struct rms_qkv_rope_append
    {
        static constexpr int opcode = OPCODE_RMS_QKV_MatVecRopeAppend; // Op index within the layer -- controls which barrier to listen to.

        static constexpr int K_BLK_START = 2048 / Globals::matvec_block_size;
        static constexpr int V_BLK_START = 2560 / Globals::matvec_block_size;
        static constexpr int EXPECTED_ARRIVAL_COUNT = 512;

        struct parsed_instruction
        {
            int layer_idx, start_block_idx, end_block_idx, iters;
            __device__ inline parsed_instruction(typename Config::instruction_t &instruction)
            {
                layer_idx = instruction[1];       // in units of 1
                start_block_idx = instruction[2]; // in units of 16 elements
                end_block_idx = instruction[3];   // in units of 16 elements
                iters = end_block_idx - start_block_idx;
            }
            __device__ inline parsed_instruction(state<Config> &s) : parsed_instruction(s.instruction()) {}
        };

        struct pipeline_specifics
        {

            static __device__ inline void gmem_wait(const Globals &g, state<Config> &s)
            {
                parsed_instruction inst{s};
                if (inst.layer_idx > 0)
                {
                    while (*(volatile int *)&g.Bar[{inst.layer_idx - 1, OPCODE_DownProjResidual - 1, 0}] < EXPECTED_ARRIVAL_COUNT)
                    {
                        __nanosleep(Config::GMEM_SPIN_LOOP_SLEEP_NANOS);
                    }
                }
            }

            static __device__ inline void store(state<Config> &s, const Globals &g, parsed_instruction &inst, int output_idx, int output_stage, semaphore &sem, int bit)
            {

                int block_idx = inst.start_block_idx + output_idx;

                // apply rope

                // even for V, we need to cast from float to bf16
                sv_fl<16> &qkv_proj_smem = *reinterpret_cast<sv_fl<16> *>((float *)s.scratch() + (32 * output_stage));
                sv_bf<16> &qkv_proj_smem_bf = *reinterpret_cast<sv_bf<16> *>((float *)s.scratch() + (32 * output_stage));

                rv_fl<16> qkv_proj, rope_cos, rope_sin;

                warp::load(rope_cos, g.rope_cos, {0, 0, static_cast<int>(g.pos_id), block_idx % 4});
                warp::load(rope_sin, g.rope_sin, {0, 0, static_cast<int>(g.pos_id), block_idx % 4});

                wait(sem, bit);
                warp::load(qkv_proj, qkv_proj_smem);

                if (block_idx < V_BLK_START)
                { // only Q & K need RoPE

                    // Fetch the neighbor values
                    int mod = (laneid() & 0b1) ? -1 : 1; // 1 for even, -1 for odd
                    warp::sync();
                    float pair_val = __shfl_sync(MASK_ALL, qkv_proj[0][0], laneid() + mod);

                    // Compute RoPE in-place
                    if (laneid() < 16)
                    {
                        // will clean this up later
                        qkv_proj[0][0] = float(qkv_proj[0][0]) * rope_cos[0][0] + float(-1 * mod) * float(pair_val) * rope_sin[0][0];
                    }
                }

                warp::store(qkv_proj_smem_bf, qkv_proj);
                warp::sync();

                if (laneid() == 0)
                {

                    if (block_idx < K_BLK_START)
                    { // Q
                        tma::store_async<cache_policy::NORMAL>(g.q_post_rope, qkv_proj_smem_bf, {0, 0, 0, block_idx});
                    }
                    else if (block_idx < V_BLK_START)
                    { // K
                        int base_index = (block_idx - K_BLK_START) * Globals::matvec_block_size;
                        int head_idx = base_index / Globals::head_dim;
                        int dim_idx = (base_index % Globals::head_dim) / Globals::matvec_block_size;
                        tma::store_async<cache_policy::NORMAL>(g.k_cache, qkv_proj_smem_bf, {inst.layer_idx, static_cast<int>(g.pos_id), head_idx, dim_idx});
                    }
                    else
                    { // V
                        int base_index = (block_idx - V_BLK_START) * Globals::matvec_block_size;
                        int head_idx = base_index / Globals::head_dim;
                        int dim_idx = (base_index % Globals::head_dim) / Globals::matvec_block_size;
                        tma::store_async<cache_policy::NORMAL>(g.v_cache, qkv_proj_smem_bf, {inst.layer_idx, static_cast<int>(g.pos_id), head_idx, dim_idx});
                    }

                    tma::store_async_wait();              // not just read wait! full wait! must be visible in global!
                    asm volatile("fence.acq_rel.gpu;\n"); // possible we need sc here but I don't think so.
                    atomicAdd(&g.Bar[{inst.layer_idx, opcode - 1, block_idx / 4}], 1);
                }

                warp::sync();
                warp::zero(qkv_proj_smem);
                warp::sync();
            }
        };

        using pipeline = rms_matvec_pipeline<Config, Globals, parsed_instruction, pipeline_specifics>;

        struct controller
        {
            static __device__ int release_lid(const Globals &g, typename Config::instruction_t &instruction, int &query)
            {
                return pipeline::release_lid(g, instruction, query);
            }
            static __device__ int init_semaphores(const Globals &g, state<Config> &s)
            {
                return pipeline::init_semaphores(s);
            }
        };
        struct loader
        {
            static __device__ void run(const Globals &g, state<Config> &s)
            {
                // Need to clear the first few elements of the scratch buffer, since we are using atomicAdd later.
                s.template zero_scratch<1024>();

                parsed_instruction inst{s};
                pipeline::loader_loop<&Globals::qkv_weights>(s, g, inst.layer_idx);
            }
        };
        struct launcher
        {
            static __device__ void run(const Globals &g, state<Config> &s)
            {

                parsed_instruction inst{s};
                pipeline::launcher_load_rms_and_activations<&Globals::hidden_states, &Globals::attn_norm_weights>(s, g, inst.layer_idx);
            }
        };
        struct consumer
        {
            static __device__ void run(const Globals &g, state<Config> &s)
            {
                pipeline::consumer_loop(s, g);
            }
        };
        struct storer
        {
            // Uses 4 full pages for outputs.
            static __device__ void run(const Globals &g, state<Config> &s)
            {
                pipeline::storer_loop(s, g);
            }
        };
    };
}
