#include "hip/hip_runtime.h"
#include "llama.cuh"
#include "utils.cuh"

using namespace kittens;
using namespace kittens::prototype;

namespace kittens::prototype::vm
{

    using globals = llama_1b_globals;

    template <typename Config, typename Globals>
    struct rms_qkv_rope_append
    {
        static constexpr int opcode = OPCODE_RMS_QKV_MatVecRopeAppend; // Op index within the layer -- controls which barrier to listen to.
        static constexpr int NUM_WEIGHT_PAGES = 4;

        static constexpr int PAGE_RMS_SCALE_ACTIVATION = 0;
        static constexpr int PAGE_ROPE_COS = PAGE_RMS_SCALE_ACTIVATION + 1;
        static constexpr int PAGE_ROPE_SIN = PAGE_ROPE_COS + 1;
        static constexpr int PAGE_WEIGHT_START = PAGE_ROPE_SIN + 1;
        static constexpr int PAGE_COUNT = PAGE_WEIGHT_START + NUM_WEIGHT_PAGES;

        static constexpr int K_BLK_START = 2048 / Globals::matvec_block_size;
        static constexpr int V_BLK_START = 2560 / Globals::matvec_block_size;

        static constexpr int REDUCTION_DIM_PER_WARP = Globals::hidden_dim / Config::NUM_CONSUMER_WARPS;

        struct parsed_instruction
        {
            int layer_idx;
            int qkv_block_idx;
            __device__ inline parsed_instruction(typename Config::instruction_t &instruction)
            {
                layer_idx = instruction[1];     // in units of 1
                qkv_block_idx = instruction[2]; // in units of 16 elements
            }
            __device__ inline parsed_instruction(state<Config> &s) : parsed_instruction(s.instruction()) {}
        };

        // Semaphores
        __device__ static inline semaphore &weights_arrived(state<Config> &s, int id) { return s.semaphores()[id]; }
        __device__ static inline semaphore &activations_arrived(state<Config> &s) { return s.semaphores()[NUM_WEIGHT_PAGES]; }
        __device__ static inline semaphore &rms_scale_arrived(state<Config> &s) { return s.semaphores()[NUM_WEIGHT_PAGES + 1]; }
        __device__ static inline semaphore &rope_cos_arrived(state<Config> &s) { return s.semaphores()[NUM_WEIGHT_PAGES + 2]; }
        __device__ static inline semaphore &rope_sin_arrived(state<Config> &s) { return s.semaphores()[NUM_WEIGHT_PAGES + 3]; }
        __device__ static inline semaphore &outputs_arrived(state<Config> &s) { return s.semaphores()[NUM_WEIGHT_PAGES + 4]; }

        // Pages (very naive for now, no fine-grained usage)
        __device__ static inline int get_rms_scale_activation_page(state<Config> &s) { return s.pid(PAGE_RMS_SCALE_ACTIVATION); }
        __device__ static inline int get_weight_page(state<Config> &s, int offset) { return s.pid(PAGE_WEIGHT_START + offset); }
        __device__ static inline int get_rope_cos_page(state<Config> &s) { return s.pid(PAGE_ROPE_COS); }
        __device__ static inline int get_rope_sin_page(state<Config> &s) { return s.pid(PAGE_ROPE_SIN); }

        struct controller
        {
            static __device__ int release_lid(const Globals &g, typename Config::instruction_t &instruction, int &query)
            {

                // unused pages, then activation, then rms scale, then weights, then rope cos, then rope sin
                int ret_order[13] = {7, 8, 9, 10, 11, 12, PAGE_RMS_SCALE_ACTIVATION, PAGE_WEIGHT_START, PAGE_WEIGHT_START + 1, PAGE_WEIGHT_START + 2, PAGE_WEIGHT_START + 3, PAGE_ROPE_COS, PAGE_ROPE_SIN};
                return ret_order[query];
            }
            static __device__ int init_semaphores(const Globals &g, state<Config> &s)
            {
                for (int i = 0; i < NUM_WEIGHT_PAGES; i++)
                {
                    init_semaphore(weights_arrived(s, i), 1);
                }

                init_semaphore(activations_arrived(s), 1);
                init_semaphore(rms_scale_arrived(s), 1);
                init_semaphore(rope_cos_arrived(s), 1);
                init_semaphore(rope_sin_arrived(s), 1);
                init_semaphore(outputs_arrived(s), 1);
                return 9;
            }
        };
        struct loader
        {
            static __device__ void run(const Globals &g, state<Config> &s)
            {
                parsed_instruction inst{s};
                // Need to clear the first few elements of the scratch buffer, since we are using atomicAdd later.
                ((uint64_t *)s.scratch())[laneid()] = 0;
                warp::sync(); // done, now we can proceed to other things.

                if (laneid() == 0)
                {

                    // RMS scale
                    int rms_scale_activation_page = get_rms_scale_activation_page(s);
                    s.wait_page_ready(rms_scale_activation_page);

                    auto &rms_scale = *reinterpret_cast<sv_bf<2048> *>(s.pages[rms_scale_activation_page].ptr());
                    s.record(TEVENT_TRIPLES_START);
                    tma::expect(rms_scale_arrived(s), rms_scale);
                    tma::load_async(rms_scale, g.attn_norm_weights, {inst.layer_idx, 0}, rms_scale_arrived(s));
                    // arrive(rms_scale_arrived(s), 1);

                    for (int i = 0; i < NUM_WEIGHT_PAGES; i++)
                    {
                        // QKV projection weights
                        auto page_id = get_weight_page(s, i);

                        s.wait_page_ready(page_id);
                        auto &weight_chunk = reinterpret_cast<st_bf<16, 512> &>(s.pages[page_id]);
                        s.record(TEVENT_TRIPLES_START + 1 + i);
                        tma::expect(weights_arrived(s, i), weight_chunk);
                        tma::load_async(weight_chunk, g.qkv_weights, {inst.layer_idx, inst.qkv_block_idx, i}, weights_arrived(s, i));
                    }

                    // Rope cos
                    auto cos_page_id = get_rope_cos_page(s);
                    s.wait_page_ready(cos_page_id);
                    auto &rope_cos = reinterpret_cast<sv_fl<16> &>(s.pages[cos_page_id]);
                    s.record(TEVENT_TRIPLES_START + 5);
                    tma::expect(rope_cos_arrived(s), rope_cos);
                    tma::load_async(rope_cos, g.rope_cos, {0, 0, static_cast<int>(g.pos_id), inst.qkv_block_idx % 4}, rope_cos_arrived(s));
                    // arrive(rope_cos_arrived(s), 1);

                    // Rope sin
                    auto sin_page_id = get_rope_sin_page(s);
                    s.wait_page_ready(sin_page_id);
                    auto &rope_sin = reinterpret_cast<sv_fl<16> &>(s.pages[sin_page_id]);
                    s.record(TEVENT_TRIPLES_START + 6);
                    tma::expect(rope_sin_arrived(s), rope_sin);
                    tma::load_async(rope_sin, g.rope_sin, {0, 0, static_cast<int>(g.pos_id), inst.qkv_block_idx % 4}, rope_sin_arrived(s));
                    // arrive(rope_sin_arrived(s), 1);
                }

                else if (laneid() >= PAGE_COUNT && laneid() < Config::NUM_PAGES)
                {
                    // Unused pages
                    auto pid = s.pid(laneid());
                    s.wait_page_ready(pid);
                    s.finish_page(pid, Config::NUM_CONSUMER_WARPS);
                }
            }
        };
        struct launcher
        {
            static __device__ void run(const Globals &g, state<Config> &s)
            {
                if (warp::laneid() == 0)
                {
                    s.wait_tensor_ready();
                    arrive(s.tensor_finished, Config::NUM_CONSUMER_WARPS);

                    parsed_instruction inst{s};

                    // Activation
                    int rms_scale_activation_page = get_rms_scale_activation_page(s);
                    s.wait_page_ready(rms_scale_activation_page);
                    auto &activations = *reinterpret_cast<sv_bf<2048> *>(s.pages[rms_scale_activation_page].ptr(sizeof(sv_bf<2048>)));

                    s.record(TEVENT_AT_GMEM_WAIT);
                    while (inst.layer_idx > 0 && *(volatile int *)&g.Bar[{inst.layer_idx - 1, OPCODE_DownProjResidual - 1, 0}] < 512)
                        __nanosleep(20);
                    s.record(TEVENT_DONE_GMEM_WAIT);
                    s.record(TEVENT_TRIPLES_START + 7);
                    tma::expect(activations_arrived(s), activations);
                    tma::load_async(activations, g.hidden_states, {}, activations_arrived(s));
                }
            }
        };
        struct consumer
        {
            static __device__ void run(const Globals &g, state<Config> &s)
            {
                // Setup
                using float_rt_t = rt_fl<16, REDUCTION_DIM_PER_WARP>;
                using float_rv_t = rv_fl<16>;

                parsed_instruction inst{s};
                typename float_rt_t::row_vec activations_vec;
                rv_fl<REDUCTION_DIM_PER_WARP> activations_vec_naive;
                float_rv_t qkv_proj, rope_cos, rope_sin;

                static_assert(Config::NUM_CONSUMER_WARPS % NUM_WEIGHT_PAGES == 0, "NUM_CONSUMER_WARPS must be divisible by NUM_WEIGHT_PAGES");
                constexpr int WARPS_PER_PAGE = Config::NUM_CONSUMER_WARPS / NUM_WEIGHT_PAGES;

                int page_index = warpid() / WARPS_PER_PAGE;

                if (group<16>::laneid() == 0)
                {
                    s.record(RMS_START);
                }

                rms_norm(g, s, activations_vec_naive, get_rms_scale_activation_page(s), activations_arrived(s), rms_scale_arrived(s), 16);

                warp::sync();
                // release the activation page
                s.warp_finish_page(get_rms_scale_activation_page(s), 1);

                if (group<16>::laneid() == 0)
                {
                    s.record(RMS_DONE);
                }

                warp::copy(activations_vec, activations_vec_naive);
                matvec<float_rt_t, WARPS_PER_PAGE>(g, s, activations_vec, weights_arrived(s, page_index), get_weight_page(s, page_index), 0);

                group<Config::NUM_CONSUMER_WARPS>::sync(1); // must wait for all warps to finish atomic add

                // release pages
                for (int i = 0; i < NUM_WEIGHT_PAGES; i++)
                {
                    s.warp_finish_page(get_weight_page(s, i), 1);
                }

                // Step 5: Apply RoPE
                if (warpid() == 0)
                { // only a single warp needed from here!

                    // even for V, we need to cast from float to bf16
                    sv_fl<16> &qkv_proj_smem = *reinterpret_cast<sv_fl<16> *>(s.scratch());
                    sv_bf<16> &qkv_proj_smem_bf = *reinterpret_cast<sv_bf<16> *>(s.scratch());
                    warp::load(qkv_proj, qkv_proj_smem);

                    warp::sync();

                    int rope_cos_page = get_rope_cos_page(s);
                    int rope_sin_page = get_rope_sin_page(s);

                    if (inst.qkv_block_idx < V_BLK_START)
                    { // only Q & K need RoPE

                        sv_fl<16> &rope_cos_smem = reinterpret_cast<sv_fl<16> &>(s.pages[rope_cos_page]);
                        wait(rope_cos_arrived(s), 0);
                        if (laneid() == 0)
                        {
                            s.record(TEVENT_TRIPLES_END + 5);
                            s.record(TEVENT_CONSUMER_START + 48);
                        }
                        warp::load(rope_cos, rope_cos_smem);
                        // warp::arrive(s.page_finished[rope_cos_page], Config::NUM_CONSUMER_WARPS);
                        s.warp_finish_page(rope_cos_page, Config::NUM_CONSUMER_WARPS);

                        sv_fl<16> &rope_sin_smem = reinterpret_cast<sv_fl<16> &>(s.pages[rope_sin_page]);
                        wait(rope_sin_arrived(s), 0);
                        if (laneid() == 0)
                        {
                            s.record(TEVENT_TRIPLES_END + 6);
                            s.record(TEVENT_CONSUMER_START + 49);
                        }
                        warp::load(rope_sin, rope_sin_smem);
                        s.warp_finish_page(rope_sin_page, Config::NUM_CONSUMER_WARPS);

                        // Fetch the neighbor values
                        int mod = (laneid() & 0b1) ? -1 : 1; // 1 for even, -1 for odd
                        warp::sync();
                        float pair_val = __shfl_sync(MASK_ALL, qkv_proj[0][0], laneid() + mod);

                        // Compute RoPE in-place
                        if (laneid() < 16)
                        {
                            // will clean this up later
                            qkv_proj[0][0] = float(qkv_proj[0][0]) * rope_cos[0][0] + float(-1 * mod) * float(pair_val) * rope_sin[0][0];
                        }
                    }
                    else
                    {
                        wait(rope_cos_arrived(s), 0);
                        s.warp_finish_page(rope_cos_page, Config::NUM_CONSUMER_WARPS);

                        wait(rope_sin_arrived(s), 0);
                        s.warp_finish_page(rope_sin_page, Config::NUM_CONSUMER_WARPS);
                    }

                    // Store back to the scratch
                    warp::store(qkv_proj_smem_bf, qkv_proj);
                    warp::sync();

                    warp::arrive(outputs_arrived(s));
                }
            }
        };
        struct storer
        {
            // Uses 4 full pages for outputs.
            static __device__ void run(const Globals &g, state<Config> &s)
            {
                if (warp::laneid() == 0)
                {
                    s.record(TEVENT_TRIPLES_STORE_START);
                }

                parsed_instruction inst{s};

                if (warp::laneid() == 0)
                {
                    sv_bf<16> &qkv_proj_smem = *reinterpret_cast<sv_bf<16> *>(s.scratch());
                    wait(outputs_arrived(s), 0);
                    s.record(TEVENT_TRIPLES_OUTPUT_READY);

                    if (inst.qkv_block_idx < K_BLK_START)
                    { // Q
                        tma::store_async<cache_policy::NORMAL>(g.q_post_rope, qkv_proj_smem, {0, 0, 0, inst.qkv_block_idx});
                    }
                    else if (inst.qkv_block_idx < V_BLK_START)
                    { // K
                        int base_index = (inst.qkv_block_idx - K_BLK_START) * Globals::matvec_block_size;
                        int head_idx = base_index / Globals::head_dim;
                        int dim_idx = (base_index % Globals::head_dim) / Globals::matvec_block_size;
                        tma::store_async<cache_policy::NORMAL>(g.k_cache, qkv_proj_smem, {inst.layer_idx, static_cast<int>(g.pos_id), head_idx, dim_idx});
                    }
                    else
                    { // V
                        int base_index = (inst.qkv_block_idx - V_BLK_START) * Globals::matvec_block_size;
                        int head_idx = base_index / Globals::head_dim;
                        int dim_idx = (base_index % Globals::head_dim) / Globals::matvec_block_size;
                        tma::store_async<cache_policy::NORMAL>(g.v_cache, qkv_proj_smem, {inst.layer_idx, static_cast<int>(g.pos_id), head_idx, dim_idx});
                    }

                    tma::store_async_wait(); // not just read wait! full wait! must be visible in global!
                    s.record(126);
                }

                warp::sync();
                asm volatile("fence.acq_rel.gpu;\n"); // possible we need sc here but I don't think so.

                if (warp::laneid() == 0)
                {
                    atomicAdd(&g.Bar[{inst.layer_idx, opcode - 1, inst.qkv_block_idx / 4}], 1);
                }
            }
        };
    };
}
