#include "hip/hip_runtime.h"
#include "llama.cuh"

using namespace kittens;
using namespace kittens::prototype;

/*
What do we need to do here... normalize entire hidden state vector
Need to prevent activation values from getting too large or small as go through layers
Calculate rms_norm for one entire hidden state 1 x LLAMA_8B_HIDDEN_DIM
*/

namespace kittens::prototype::vm
{

    using globals = llama_8b_globals;

    template <
        auto weights_ptr,
        auto outputs_ptr,
        int _opcode,
        typename Config = kittens::prototype::vm::default_config>
    struct rms_op
    {
        static constexpr int opcode = _opcode;
        static constexpr int REDUCTION_DIM_PER_WARP = globals::hidden_dim / Config::NUM_CONSUMER_WARPS;

        struct parsed_instruction
        {
            int layer_idx;
            int batch_idx;
            __device__ inline parsed_instruction(typename Config::instruction_t &instruction)
            {
                layer_idx = instruction[1];
                batch_idx = instruction[2];
            }
            __device__ inline parsed_instruction(state<Config> &s) : parsed_instruction(s.instruction()) {}
        };

        // Semaphores
        __device__ static inline semaphore &activations_arrived(state<Config> &s) { return s.semaphores()[0]; }
        __device__ static inline semaphore &weights_arrived(state<Config> &s) { return s.semaphores()[1]; }
        __device__ static inline semaphore &outputs_arrived(state<Config> &s) { return s.semaphores()[2]; }

        // Pages (very naive for now, no fine-grained usage)
        static constexpr int PAGE_WEIGHT = 0;
        static constexpr int PAGE_ACTIVATION = 1;
        __device__ static inline int get_weight_page(state<Config> &s) { return s.pid(PAGE_WEIGHT); }
        __device__ static inline int get_activation_page(state<Config> &s) { return s.pid(PAGE_ACTIVATION); }

        struct controller
        {
            static __device__ int release_lid(const globals &g, typename Config::instruction_t &instruction, int &query)
            {

                return query;
            }
            static __device__ int init_semaphores(const globals &g, state<Config> &s)
            {
                init_semaphore(activations_arrived(s), 1);
                init_semaphore(weights_arrived(s), 1);
                init_semaphore(outputs_arrived(s), Config::NUM_CONSUMER_WARPS);
                return 3;
            }
        };
        struct loader
        {
            static __device__ inline void gmem_wait(const globals &g, state<Config> &s) {}

            static __device__ void run(const globals &g, state<Config> &s)
            {
                if (warp::laneid() == 0)
                {
                    s.record(TEVENT_LOADER_START);
                }
                parsed_instruction inst{s};
                // Need to clear the first few elements of the scratch buffer, since we are using atomicAdd later.
                ((uint64_t *)s.scratch())[laneid()] = 0;
                warp::sync(); // done, now we can proceed to other things.

                if (laneid() == 0)
                {
                    // RMS scale
                    int weight_page = get_weight_page(s);
                    s.wait_page_ready(weight_page);
                    auto &rms_scale = *reinterpret_cast<sv_bf<globals::hidden_dim> *>(s.pages[weight_page].ptr());

                    tma::expect(weights_arrived(s), rms_scale);
                    auto &weights_global = g.*weights_ptr;
                    tma::load_async(rms_scale, weights_global, {inst.layer_idx, 0}, weights_arrived(s));

                    // Activation
                    int act_page = get_activation_page(s);
                    s.wait_page_ready(act_page);
                    s.record(TEVENT_AT_GMEM_WAIT);

                    gmem_wait(g, s);

                    s.record(TEVENT_DONE_GMEM_WAIT);
                    auto &activations = *reinterpret_cast<sv_bf<globals::hidden_dim> *>(s.pages[act_page].ptr());

                    tma::expect(activations_arrived(s), activations);
                    tma::load_async(activations, g.hidden_states, {inst.batch_idx, 0}, activations_arrived(s));
                }

                else if (laneid() >= 2 && laneid() <= 12)
                {
                    // Unused pages
                    s.wait_page_ready(s.pid(laneid()));
                    arrive(s.page_finished[s.pid(laneid())][0], Config::NUM_CONSUMER_WARPS);
                }

                warp::sync();
                if (warp::laneid() == 0)
                {
                    s.record(TEVENT_LOADER_END);
                }
            }
        };
        struct launcher
        {
            static __device__ void run(const globals &g, state<Config> &s)
            {
                if (warp::laneid() == 0)
                {
                    s.wait_tensor_ready();
                    arrive(s.tensor_finished, Config::NUM_CONSUMER_WARPS);
                }
            }
        };
        struct consumer
        {
            static __device__ void run(const globals &g, state<Config> &s)
            {
                if (warp::laneid() == 0)
                {
                    s.record(TEVENT_CONSUMER_START + warpid());
                }

                // Setup
                parsed_instruction inst{s};
                rv_fl<REDUCTION_DIM_PER_WARP> activations_vec, copy_activations_vec, rms_scale_vec;
                sv_bf<REDUCTION_DIM_PER_WARP> *rms_scale_smem = reinterpret_cast<sv_bf<REDUCTION_DIM_PER_WARP> *>(s.pages[get_weight_page(s)].ptr());
                sv_bf<REDUCTION_DIM_PER_WARP> *activations_smem = reinterpret_cast<sv_bf<REDUCTION_DIM_PER_WARP> *>(s.pages[get_activation_page(s)].ptr());

                // Setup
                wait(activations_arrived(s), 0);

                warp::load(activations_vec, activations_smem[warpid()]);
                warp::sync();

                // Step 2: Apply RMS normalization
                warp::copy(copy_activations_vec, activations_vec);                           // cast to float
                warp::mul(copy_activations_vec, copy_activations_vec, copy_activations_vec); // square
                float partial_sum = warp::sum(copy_activations_vec);

                auto smem_rms_partial_sums = ((float *)s.scratch());
                // aggregate sums across the consumer warps
                if (laneid() == 0)
                {
                    smem_rms_partial_sums[warpid()] = partial_sum;
                }

                group<Config::NUM_CONSUMER_WARPS>::sync(0);

                float full_sum = 0;
                for (int i = 0; i < Config::NUM_CONSUMER_WARPS; i++)
                {
                    full_sum += smem_rms_partial_sums[i];
                }

                float variance = full_sum / (float)globals::hidden_dim;
                float rms_scale = rsqrtf(variance + g.rms_norm_eps);

                warp::copy(copy_activations_vec, activations_vec); // unsquare
                warp::mul(copy_activations_vec, copy_activations_vec, rms_scale);
                warp::copy(activations_vec, copy_activations_vec);

                // multiply by rms scale
                wait(weights_arrived(s), 0);

                warp::load(rms_scale_vec, rms_scale_smem[warpid()]);
                warp::sync();

                warp::mul(activations_vec, activations_vec, rms_scale_vec);

                // Need to ensure storing here is correct!!!
                warp::store(activations_smem[warpid()], activations_vec);
                warp::sync();
                warp::arrive(outputs_arrived(s));
            }
        };
        struct storer
        {
            // Uses 4 full pages for outputs.
            static __device__ void run(const globals &g, state<Config> &s)
            {
                if (warp::laneid() == 0)
                {
                    s.record(TEVENT_TRIPLES_STORE_START);
                }

                parsed_instruction inst{s};

                if (warp::laneid() == 0)
                {
                    wait(outputs_arrived(s), 0);
                    int activation_page = get_activation_page(s);
                    auto &rms_activations = *reinterpret_cast<sv_bf<globals::hidden_dim> *>(s.pages[activation_page].ptr());
                    auto &outputs_global = g.*outputs_ptr;
                    tma::store_async<cache_policy::NORMAL>(outputs_global, rms_activations, {inst.batch_idx, 0});
                    tma::store_async_wait();

                    s.finish_page(activation_page, Config::NUM_CONSUMER_WARPS);
                    s.finish_page(get_weight_page(s), Config::NUM_CONSUMER_WARPS);
                }

                warp::sync();
                asm volatile("fence.acq_rel.gpu;\n"); // possible we need sc here but I don't think so.

                if (warp::laneid() == 0) {
                    auto batch_block_idx = inst.batch_idx / globals::matmul_batch_block_size;
                    atomicAdd(&g.Bar[{inst.layer_idx, opcode - 1, batch_block_idx, 0}], 1);
                }

                warp::sync();
                if (laneid() == 0)
                    s.record(TEVENT_STORE_END);
            }
        };
    };

    template <typename Config, typename globals>
    struct attn_norm : rms_op<&globals::attn_norm_weights, &globals::rms_rope_intermediates, OPCODE_AttnNorm, Config>
    {
        using base_op = rms_op<&globals::attn_norm_weights, &globals::rms_rope_intermediates, OPCODE_AttnNorm, Config>;
        struct loader : base_op::loader
        {
            static __device__ inline void gmem_wait(const globals &g, state<Config> &s)
            {
                typename base_op::parsed_instruction inst{s};
                auto batch_block_idx = inst.batch_idx / globals::matmul_batch_block_size;
                if (inst.layer_idx > 0)
                {
                    while (*(volatile int *)&g.Bar[{inst.layer_idx - 1, OPCODE_DownProjResidual - 1, batch_block_idx, 0}] < globals::num_output_blocks)
                    {
                        __nanosleep(20);
                    }
                }
            }
        };
    };

    template <typename Config, typename globals>
    struct mlp_norm : rms_op<
                          &globals::mlp_norm_weights,
                          &globals::rms_gate_intermediates,
                          OPCODE_MlpNorm,
                          Config>
    {
        using base_op = rms_op<&globals::mlp_norm_weights, &globals::rms_gate_intermediates, OPCODE_MlpNorm, Config>;
        struct loader : base_op::loader
        {
            static __device__ inline void gmem_wait(const globals &g, state<Config> &s)
            {
                typename base_op::parsed_instruction inst{s};
                auto batch_block_idx = inst.batch_idx / globals::matmul_batch_block_size;
                while (*(volatile int *)&g.Bar[{inst.layer_idx, OPCODE_O_ProjResidual - 1, batch_block_idx, 0}] < globals::num_output_blocks)
                {
                    __nanosleep(20);
                }
            }
        };
    };

    template <typename Config, typename globals>
    struct lm_head_norm : rms_op<
                              &globals::lm_head_norm_weights,
                              &globals::rms_lm_head_intermediates,
                              OPCODE_LM_HeadNorm,
                              Config>
    {
        using base_op = rms_op<&globals::lm_head_norm_weights, &globals::rms_lm_head_intermediates, OPCODE_LM_HeadNorm, Config>;
        struct loader : base_op::loader
        {
            static __device__ inline void gmem_wait(const globals &g, state<Config> &s)
            {
                typename base_op::parsed_instruction inst{s};
                auto batch_block_idx = inst.batch_idx / globals::matmul_batch_block_size;
                while (*(volatile int *)&g.Bar[{globals::num_hidden_layers - 1, OPCODE_DownProjResidual - 1, batch_block_idx, 0}] < globals::num_output_blocks)
                {
                    __nanosleep(20);
                }
            }
        };
    };

}
