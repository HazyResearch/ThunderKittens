#include "hip/hip_runtime.h"
#include "conversions.cuh"

#ifdef TEST_GROUP_SHARED_VEC_CONVERSIONS

struct vec_copy {
    template<int S, int NW> using valid = std::bool_constant<S%NW==0 && S<=64>; // this is group-level
    static inline const std::string test_identifier = "shared_vec_convert";
    template<int S, int NW, gl_t GL> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int S, int NW, gl_t GL> __device__ static void device_func(const GL &input, const GL &output) {
        using G = kittens::group<NW>;
        __shared__ kittens::col_vec<kittens::st_bf<16*S, 16*S>> vec1;
        __shared__ kittens::col_vec<kittens::st_bf<16*S, 16*S>> vec2;
        G::load(vec1, input, {});
        G::sync(0);
        G::copy(vec2, vec1);
        G::sync(0);
        G::store(output, vec2, {});
    }
};

void group::shared::vec::conversions::tests(test_data &results) {
    std::cout << " ----- Starting ops/group/shared/vec/conversions tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
                         
    sweep_size_1d<vec_copy, SIZE, 2>::run(results);
    sweep_size_1d<vec_copy, SIZE, 4>::run(results);
    sweep_size_1d<vec_copy, SIZE, 12>::run(results);
    std::cout << std::endl;
}

#endif