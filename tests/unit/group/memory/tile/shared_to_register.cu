#include "hip/hip_runtime.h"
#include "shared_to_register.cuh"

#ifdef TEST_GROUP_MEMORY_TILE_SHARED_TO_REGISTER

template<typename T>
struct group_shared_reg_load_store {
    using dtype = T;
    template<int H, int W, int NW, kittens::ducks::rt_layout::all RL> using valid = std::bool_constant<
        ( H%NW==0 && W*H<=64 ) 
        #ifdef KITTENS_HOPPER
        && ( (!std::is_same_v<T, kittens::fp8e4m3> && !std::is_same_v<T, kittens::fp8e5m2>) || W%2 == 0 )
        #endif
    >;
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "group_shared_reg_loadstore_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "group_shared_reg_loadstore_gmem=half" :
#ifdef KITTENS_HOPPER
                                                      std::is_same_v<T, kittens::fp8e4m3> ? "group_shared_reg_loadstore_gmem=fp8e4m3" :
                                                      std::is_same_v<T, kittens::fp8e5m2> ? "group_shared_reg_loadstore_gmem=fp8e5m2" :
#endif 
                                                                                         "group_shared_reg_loadstore_gmem=float";
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all RL> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all RL> __device__ static void device_func(const GL &input, const GL &output) {
        using G = kittens::group<NW>;
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<16> al((int*)&__shm[0]); 
        kittens::st<dtype, 16*H, 16*W> &shared_tile = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        G::load(shared_tile, input, {});
        __syncthreads();
        kittens::rt<dtype, 16*H/NW, 16*W, RL> reg_tile;
        G::load(reg_tile, shared_tile);
        __syncthreads();
        G::store(shared_tile, reg_tile);
        __syncthreads();
        G::store(output, shared_tile, {});
    }
};

void group::memory::tile::shared_to_register::tests(test_data &results) {
    std::cout << " ----- Starting ops/group/memory/tile/shared_to_register tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    sweep_size_2d<group_shared_reg_load_store<float>, SIZE, SIZE, 1, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<float>, SIZE, SIZE, 1, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<float>, SIZE, SIZE, 2, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<float>, SIZE, SIZE, 2, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<float>, SIZE, SIZE, 4, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<float>, SIZE, SIZE, 4, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<float>, SIZE, SIZE, 12, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<float>, SIZE, SIZE, 12, kittens::ducks::rt_layout::col>::run(results);

    sweep_size_2d<group_shared_reg_load_store<kittens::bf16>, SIZE, SIZE, 1, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::bf16>, SIZE, SIZE, 1, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::bf16>, SIZE, SIZE, 2, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::bf16>, SIZE, SIZE, 2, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::bf16>, SIZE, SIZE, 4, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::bf16>, SIZE, SIZE, 4, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::bf16>, SIZE, SIZE, 12, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::bf16>, SIZE, SIZE, 12, kittens::ducks::rt_layout::col>::run(results);

    sweep_size_2d<group_shared_reg_load_store<kittens::half>, SIZE, SIZE, 1, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::half>, SIZE, SIZE, 1, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::half>, SIZE, SIZE, 2, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::half>, SIZE, SIZE, 2, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::half>, SIZE, SIZE, 4, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::half>, SIZE, SIZE, 4, kittens::ducks::rt_layout::col>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::half>, SIZE, SIZE, 12, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::half>, SIZE, SIZE, 12, kittens::ducks::rt_layout::col>::run(results);

    #ifdef KITTENS_HOPPER
    sweep_size_2d<group_shared_reg_load_store<kittens::fp8e4m3>, SIZE, SIZE, 1, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::fp8e4m3>, SIZE, SIZE, 2, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::fp8e4m3>, SIZE, SIZE, 4, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::fp8e4m3>, SIZE, SIZE, 12, kittens::ducks::rt_layout::row>::run(results);

    sweep_size_2d<group_shared_reg_load_store<kittens::fp8e5m2>, SIZE, SIZE, 1, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::fp8e5m2>, SIZE, SIZE, 2, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::fp8e5m2>, SIZE, SIZE, 4, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d<group_shared_reg_load_store<kittens::fp8e5m2>, SIZE, SIZE, 12, kittens::ducks::rt_layout::row>::run(results);
    #endif
    std::cout << std::endl;
}

#endif