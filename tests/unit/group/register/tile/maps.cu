#include "hip/hip_runtime.h"
#include "maps.cuh"

#ifdef TEST_GROUP_REG_TILE_MAPS

struct test_exp {
    template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && W*H<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_exp";
    template<int H, int W, int NW, kittens::ducks::gl::all GL, kittens::ducks::rt_layout::all L> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        for(int i = 0; i < i_ref.size(); i++) o_ref[i] = ::expf(i_ref[i]);
    }
    template<int H, int W, int NW, kittens::ducks::gl::all GL, kittens::ducks::rt_layout::all L> __device__ static void device_func(const GL input, const GL output) {
        kittens::rt_bf<16*H, 16*W, L> reg_tile;
        kittens::warp::load(reg_tile, input, {});
        kittens::warp::exp(reg_tile, reg_tile);
        kittens::warp::store(output, reg_tile, {});
    }
};
struct test_tril {
    template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && H==W && W*H<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_tril";
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // triangular lower, with diagonal starting at row_idx 4
        for(int i = 0; i < H*16; i++)
            for(int j = 0; j < W*16; j++)
                o_ref[i*W*16 + j] = j<=i+(4*H) ? i_ref[i*W*16 + j] : 0;
    }
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __device__ static void device_func(const GL input, const GL output) {
        kittens::rt_fl<16*H, 16*W, L> reg_tile;
        kittens::warp::load(reg_tile, input, {});
        kittens::warp::tril(reg_tile, reg_tile, 4*H);
        kittens::warp::store(output, reg_tile, {});
    }
};
struct test_triu {
    template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && H==W && W*H<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_triu";
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // triangular upper, with diagonal starting at row_idx 4
        for(int i = 0; i < H*16; i++)
            for(int j = 0; j < W*16; j++)
                o_ref[i*W*16 + j] = j>=i+(4*H) ? i_ref[i*W*16 + j] : 0;
    }
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __device__ static void device_func(const GL input, const GL output) {
        kittens::rt_fl<16*H, 16*W, L> reg_tile;
        kittens::warp::load(reg_tile, input, {});
        kittens::warp::triu(reg_tile, reg_tile, 4*H);
        kittens::warp::store(output, reg_tile, {});
    }
};
struct test_right_fill {
    template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && H==W && W*H<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_right_fill";
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // here, set everything to from and right of col_idx 8 is set to zero
        for(int i = 0; i < H*16; i++) 
            for(int j = 0; j < W*16; j++) 
            o_ref[i*W*16 + j] = (j < (8 * W)) ? i_ref[i*W*16 + j] : 0;
    }
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __device__ static void device_func(const GL input, const GL output) {
        kittens::rt_fl<16*H, 16*W, L> reg_tile;
        kittens::warp::load(reg_tile, input, {});
        kittens::warp::apply(reg_tile, reg_tile, []__device__(int r, int c, const float &x) { return (c < (8 * W)) ? x : 0; });
        kittens::warp::store(output, reg_tile, {});
    }
};
struct test_left_fill {
    template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && H==W && W*H<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_left_fill";
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // here, set everything to from and left of col_idx 8 is set to zero
        for(int i = 0; i < H*16; i++) 
            for(int j = 0; j < W*16; j++) 
                o_ref[i*W*16 + j] = (j >= (8 * W)) ? i_ref[i*W*16 + j] : 0;
    }
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __device__ static void device_func(const GL input, const GL output) {
        kittens::rt_fl<16*H, 16*W, L> reg_tile;
        kittens::warp::load(reg_tile, input, {});
        kittens::warp::apply(reg_tile, reg_tile, []__device__(int r, int c, const float &x) { return (c >= (8 * W)) ? x : 0; });
        kittens::warp::store(output, reg_tile, {});
    }
};
struct test_lower_fill {
    template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && H==W && W*H<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_lower_fill";
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // here, set everything to from and lower of row_idx 8 is set to zero
        for(int i = 0; i < H*16; i++) 
            for(int j = 0; j < W*16; j++) 
                o_ref[i*W*16 + j] = (i < (8 * H)) ? i_ref[i*W*16 + j] : 0;
    }
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __device__ static void device_func(const GL input, const GL output) {
        kittens::rt_fl<16*H, 16*W, L> reg_tile;
        kittens::warp::load(reg_tile, input, {});
        kittens::warp::apply(reg_tile, reg_tile, []__device__(int r, int c, const float &x) { return (r < (8 * H)) ? x : 0; });
        kittens::warp::store(output, reg_tile, {});
    }
};
struct test_upper_fill {
    template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && H==W && W*H<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_upper_fill";
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        // here, set everything to from and upper of row_idx 8 is set to zero
        for(int i = 0; i < H*16; i++) 
            for(int j = 0; j < W*16; j++) 
                o_ref[i*W*16 + j] = (i >= ((8 * H))) ? i_ref[i*W*16 + j] : 0;
    }
    template<int H, int W, int NW, gl_t GL, kittens::ducks::rt_layout::all L> __device__ static void device_func(const GL input, const GL output) {
        kittens::rt_fl<16*H, 16*W, L> reg_tile;
        kittens::warp::load(reg_tile, input, {});
        kittens::warp::apply(reg_tile, reg_tile, []__device__(int r, int c, const float &x) { return (r >= (8 * H)) ? x : 0; });
        kittens::warp::store(output, reg_tile, {});
    }
};

void group::reg::tile::maps::tests(test_data &results) {
    std::cout << " ----- Starting ops/group/register/tile/maps tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    sweep_size_2d_warp<test_exp, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<test_exp, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);

    sweep_size_2d_warp<test_tril, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<test_tril, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);

    sweep_size_2d_warp<test_triu, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<test_triu, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);

    sweep_size_2d_warp<test_right_fill, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<test_right_fill, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);

    sweep_size_2d_warp<test_left_fill, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<test_left_fill, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);

    sweep_size_2d_warp<test_lower_fill, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<test_lower_fill, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);

    sweep_size_2d_warp<test_upper_fill, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<test_upper_fill, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);

    std::cout << std::endl;
}

#endif