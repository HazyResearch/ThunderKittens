#include "hip/hip_runtime.h"
#include "maps.cuh"

#ifdef TEST_WARP_REGISTER_TILE_MAPS

struct test_exp {
    template<int H, int W, int NW, kittens::ducks::rt_layout::all L> using valid = std::bool_constant<NW == 1 && W*H<=64>; // this is warp-level
    static inline const std::string test_identifier = "reg_exp";
    template<int H, int W, int NW, kittens::ducks::gl::all GL, kittens::ducks::rt_layout::all L> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        for(int i = 0; i < i_ref.size(); i++) o_ref[i] = ::expf(i_ref[i]);
    }
    template<int H, int W, int NW, kittens::ducks::gl::all GL, kittens::ducks::rt_layout::all L> __device__ static void device_func(const GL input, const GL output) {
        kittens::rt_bf<16*H, 16*W, L> reg_tile;
        kittens::warp::load(reg_tile, input, {});
        kittens::exp(reg_tile, reg_tile);
        kittens::warp::store(output, reg_tile, {});
    }
};

void warp::reg::tile::maps::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/register/tile/maps tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
    sweep_size_2d_warp<test_exp, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<test_exp, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);
}

#endif