#include "hip/hip_runtime.h"
#include "dsmem.cuh"
// #include <hip/hip_cooperative_groups.h>

#ifdef TEST_WARP_MEMORY_TILE_DSMEM

template<typename T>
struct test_dsmem { // load with dsmem, write out normally
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 && W*H*sizeof(dtype)*256*2<=kittens::MAX_SHARED_MEMORY-8192>;
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "dsmem_transfer_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "dsmem_transfer_gmem=half" :
                                                                                         "dsmem_transfer_gmem=float";
    template<int H, int W, int NW, kittens::ducks::gl::all GL> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        for(int i = 0; i < 4; i++) {
            for(int j = 0; j < H*W*256; j++) {
                o_ref[i*H*W*256 + j] = i_ref[((i+1)%4)*H*W*256 + j];
            }
        }
    }
    template<int H, int W, int NW, kittens::ducks::gl::all GL>
    __device__ static void device_func(const GL input, const GL output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_swizzle_allocator al((int*)&__shm[0]); 
        kittens::st<dtype, 16*H, 16*W> (&src_tile) = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        kittens::st<dtype, 16*H, 16*W> (&dst_tile) = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        
        kittens::warp::load(src_tile, input, kittens::coord{0, (int)blockIdx.x, 0, 0});

        __shared__ kittens::semaphore dsmem_semaphore;
        kittens::warp::init_semaphore(dsmem_semaphore, 0, 1);
        kittens::warp::tma::expect(dsmem_semaphore, dst_tile);
        kittens::everyone::tma::cluster::sync();
        kittens::warp::tma::cluster::store_async(dst_tile, src_tile, (blockIdx.x+3)%4, dsmem_semaphore);
        kittens::wait(dsmem_semaphore, 0);

        kittens::warp::store(output, dst_tile, kittens::coord{0, (int)blockIdx.x, 0, 0});
    }
};

template<typename Ker, typename T, int H, int W, int NW, kittens::ducks::gl::all GL, typename... args>
static __global__ __cluster_dims__(4, 1, 1) void dsmem_global_wrapper_2d(const GL input, const GL output) {
    Ker::template device_func<H, W, NW, GL, args...>(input, output);
}
template<typename test, int H, int W, int NUM_WORKERS, typename... args>
struct dsmem_wrapper_2d {
    using dtype = gmem_dtype<test>;
    static void run(test_data& results) {
        test_info this_result;
        this_result.label = generate_test_name<H,W,NUM_WORKERS, args...>(test::test_identifier);
        if constexpr (test::template valid<H, W, NUM_WORKERS, args...>::value) {
            constexpr int D = 4;
            constexpr int SIZE = H*W*256 * D; // D for additional dsmem cluster dimension
            // initialize
            dtype *d_i, *d_o;
            std::vector<float> i_ref(SIZE);
            std::vector<float> o_ref(SIZE);
            initialize<dtype, initializers::ARANGE>(&d_i, &d_o, i_ref, o_ref);
            // make descriptors
            using GL = typename kittens::gl<dtype, 1, D, H*16, W*16>;
            GL input(d_i, nullptr, nullptr, nullptr, nullptr);
            GL output(d_o, nullptr, nullptr, nullptr, nullptr);
            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                dsmem_global_wrapper_2d<test), dtype, H, W, NUM_WORKERS, GL, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );
            dsmem_global_wrapper_2d<test, dtype, H, W, NUM_WORKERS, GL, args...><<<4, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(input, output);
            // fill in correct results on cpu
            test::template host_func<H, W, NUM_WORKERS, GL, args...>(i_ref, o_ref);
            // check and cleanup
            this_result.result = validate(d_i, d_o, i_ref, o_ref, this_result.label, W*16);
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};
template<typename test, int MAX_H=8, int MAX_W=8, int NUM_WORKERS=1, typename... args>
using dsmem_sweep_size_2d = loop_h<dsmem_wrapper_2d, test, MAX_H, MAX_W, NUM_WORKERS, MAX_H, args...>;
template<template<typename> typename test, int MAX_H=8, int MAX_W=8, int NUM_WORKERS=1, typename... args>
struct dsmem_sweep_gmem_type_2d {
    static void run(test_data &results) {
        dsmem_sweep_size_2d<test<float>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
        dsmem_sweep_size_2d<test<kittens::bf16>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
        dsmem_sweep_size_2d<test<kittens::half>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
    }
};
template<template<typename> typename test, int MAX_H=8, int MAX_W=8, typename... args> using dsmem_sweep_gmem_type_2d_warp = dsmem_sweep_gmem_type_2d<test, MAX_H, MAX_W, 1, args...>;


void warp::memory::tile::dsmem::tests(test_data &results) {
    std::cout << " ----- Starting ops/warp/memory/tile/dsmem tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    dsmem_sweep_gmem_type_2d_warp<test_dsmem, SIZE, SIZE>::run(results);
    std::cout << std::endl;
}

#endif