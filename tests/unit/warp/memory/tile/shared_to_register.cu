#include "hip/hip_runtime.h"
#include "shared_to_register.cuh"

#ifdef TEST_WARP_MEMORY_TILE_SHARED_TO_REGISTER

template<typename T>
struct sharedreg_load_store {
    using dtype = T;
    template<int H, int W, int NW, kittens::ducks::rt_layout::all RL> using valid = std::bool_constant<
      ( NW == 1 && W*H<=64 ) 
      #ifdef KITTENS_HOPPER
      && ( (!std::is_same_v<T, kittens::fp8e4m3> && !std::is_same_v<T, kittens::fp8e5m2> ) || W%2==0 ) && 
      ( (!std::is_same_v<T, kittens::fp8e4m3> && !std::is_same_v<T, kittens::fp8e5m2> ) || std::is_same_v<RL, kittens::ducks::rt_layout::row> ) 
      #endif
    >;
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "shared_reg_loadstore_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "shared_reg_loadstore_gmem=half" :
                                                      #ifdef KITTENS_HOPPER
                                                      std::is_same_v<T, kittens::fp8e4m3> ? "shared_reg_loadstore_gmem=fp8e4m3" :
                                                      std::is_same_v<T, kittens::fp8e5m2> ? "shared_reg_loadstore_gmem=fp8e5m2" :
                                                      #endif
                                                                                         "shared_reg_loadstore_gmem=float";
    template<int H, int W, int NW, kittens::ducks::gl::all GL, kittens::ducks::rt_layout::all RL> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        o_ref = i_ref; // overwrite the whole thing
    }
    template<int H, int W, int NW, kittens::ducks::gl::all GL, kittens::ducks::rt_layout::all RL> __device__ static void device_func(const GL input, const GL output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::shared_allocator<16> al((int*)&__shm[0]); 
        kittens::st<T, 16*H, 16*W> &shared_tile = al.allocate<kittens::st<T, 16*H, 16*W>>();
        kittens::warp::load(shared_tile, input, {0, 0, 0, 0});
        __syncthreads();
        kittens::rt<T, 16*H, 16*W, RL> reg_tile;
        kittens::warp::load(reg_tile, shared_tile);
        __syncthreads();
        kittens::warp::store(shared_tile, reg_tile);
        __syncthreads();
        kittens::warp::store(output, shared_tile, {0, 0, 0, 0});
    }
};

void warp::memory::tile::shared_to_register::tests(test_data &results) {
    std::cout << "\n ----- Starting ops/warp/memory/tile/shared_to_register tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    sweep_gmem_type_2d_warp<sharedreg_load_store, SIZE, SIZE, kittens::ducks::rt_layout::row>::run(results);
    sweep_gmem_type_2d_warp<sharedreg_load_store, SIZE, SIZE, kittens::ducks::rt_layout::col>::run(results);
}

#endif