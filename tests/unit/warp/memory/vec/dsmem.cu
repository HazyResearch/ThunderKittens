#include "hip/hip_runtime.h"
#include "dsmem.cuh"

#ifdef TEST_WARP_MEMORY_VEC_DSMEM

template<typename T>
struct test_dsmem_vec { // load with dsmem, write out normally
    using dtype = T;
    template<int S, int NW> using valid = std::bool_constant<NW == 1>; // note the 128 byte multiple requirement
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "dsmem_vec_transfer_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "dsmem_vec_transfer_gmem=half" :
                                                                                         "dsmem_vec_transfer_gmem=float";
    template<int S, int NW, kittens::ducks::gl::all GL> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        for(int i = 0; i < 4; i++) {
            for(int j = 0; j < S*16; j++) {
                o_ref[i*S*16 + j] = i_ref[((i+1)%4)*S*16 + j];
            }
        }
    }
    template<int S, int NW, kittens::ducks::gl::all GL>
    __device__ static void device_func(const GL &input, const GL &output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_swizzle_allocator al((int*)&__shm[0]); 
        kittens::row_vec<kittens::st<dtype, 16*S, 16*S>> (&src_vec) = al.allocate<kittens::row_vec<kittens::st<dtype, 16*S, 16*S>>>();
        kittens::row_vec<kittens::st<dtype, 16*S, 16*S>> (&dst_vec) = al.allocate<kittens::row_vec<kittens::st<dtype, 16*S, 16*S>>>();

        __shared__ kittens::semaphore dsmem_semaphore;
        kittens::warp::load(src_vec, input, {(int)blockIdx.x, 0});

        kittens::warp::init_semaphore(dsmem_semaphore, 0, 1);
        kittens::warp::tma::expect(dsmem_semaphore, dst_vec);

        kittens::everyone::tma::cluster::sync();

        kittens::warp::tma::cluster::store_async(dst_vec, src_vec, (blockIdx.x+3)%4, dsmem_semaphore);

        kittens::wait(dsmem_semaphore, 0);

        kittens::warp::store(output, dst_vec, {(int)blockIdx.x, 0});
    }
};

template<typename Ker, typename T, int S, int NW, kittens::ducks::gl::all GL, typename... args>
static __global__ __cluster_dims__(4, 1, 1) void dsmem_global_wrapper_1d(GL input, const GL output) {
    Ker::template device_func<S, NW, GL, args...>(input, output);
}
template<typename test, int S, int NUM_WORKERS, typename... args>
struct dsmem_wrapper_1d {
    using dtype = gmem_dtype<test>;
    static void run(test_data& results) {
        test_info this_result;
        this_result.label = generate_test_name<S, NUM_WORKERS, args...>(test::test_identifier);
        if constexpr (test::template valid<S, NUM_WORKERS, args...>::value) {
            constexpr int SIZE = S*16 * 4; // 4 for additional dsmem cluster dimension
            // initialize
            dtype *d_i, *d_o;
            std::vector<float> i_ref(SIZE);
            std::vector<float> o_ref(SIZE);
            initialize<dtype, initializers::ARANGE>(&d_i, &d_o, i_ref, o_ref);
            // make descriptors
            using GL = typename kittens::gl<dtype, 1, 1, 4, 16*S>;
            GL input(d_i, nullptr, nullptr, nullptr, nullptr);
            GL output(d_o, nullptr, nullptr, nullptr, nullptr);
            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                dsmem_global_wrapper_1d<test), dtype, S, NUM_WORKERS, GL, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );
            dsmem_global_wrapper_1d<test, dtype, S, NUM_WORKERS, GL, args...><<<4, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(input, output);
            // fill in correct results on cpu
            test::template host_func<S, NUM_WORKERS, GL, args...>(i_ref, o_ref);
            // check and cleanup
            this_result.result = validate(d_i, d_o, i_ref, o_ref, this_result.label, S*16);
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};
template<typename test, int MAX_S, int NW, typename... args>
using dsmem_sweep_size_1d = loop_s<dsmem_wrapper_1d, test, MAX_S, NW, MAX_S, args...>;
template<template<typename> typename test, int MAX_S=8, int NUM_WORKERS=1, typename... args>
struct dsmem_sweep_gmem_type_1d {
    static void run(test_data &results) {
        dsmem_sweep_size_1d<test<float>, MAX_S, NUM_WORKERS, args...>::run(results);
        dsmem_sweep_size_1d<test<kittens::bf16>, MAX_S, NUM_WORKERS, args...>::run(results);
        dsmem_sweep_size_1d<test<kittens::half>, MAX_S, NUM_WORKERS, args...>::run(results);
    }
};
template<template<typename> typename test, int MAX_S=8, typename... args> using dsmem_sweep_gmem_type_1d_warp = dsmem_sweep_gmem_type_1d<test, MAX_S, 1, args...>;

void warp::memory::vec::dsmem::tests(test_data &results) {
    std::cout << " ----- Starting ops/warp/memory/vec/dsmem tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    dsmem_sweep_gmem_type_1d_warp<test_dsmem_vec, SIZE>::run(results);
    std::cout << std::endl;
}

#endif