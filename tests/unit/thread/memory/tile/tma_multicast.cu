#include "hip/hip_runtime.h"
#include "tma_multicast.cuh"

#ifdef TEST_THREAD_MEMORY_TILE_TMA_MULTICAST

template<typename T>
struct test_load_multicast { // load with TMA, write out normally
    using dtype = T;
    template<int H, int W, int NW> using valid = std::bool_constant<NW == 1 && W*H*sizeof(dtype)*256<=kittens::MAX_SHARED_MEMORY-4096>; // S%4 ensures alignment
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "tma_multicast_load_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "tma_multicast_load_gmem=half" :
                                                                                         "tma_multicast_load_gmem=float";
    template<int H, int W, int NW, kittens::ducks::gl::all GL> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        int SIZE_DIV_4 = i_ref.size()/4;
        for(int i = 0; i < SIZE_DIV_4; i++) {
            for(int j = 0; j < 4; j++) {
                o_ref[i+j*SIZE_DIV_4] = i_ref[i];
            }
        }
    }
    template<int H, int W, int NW, kittens::ducks::gl::all GL>
    __device__ static void device_func(const GL &input, const GL &output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_swizzle_allocator al((int*)&__shm[0]); 
        kittens::st<dtype, 16*H, 16*W> (&shared_tile) = al.allocate<kittens::st<dtype, 16*H, 16*W>>();
        int rank = blockIdx.x % 4;
        
        __shared__ kittens::semaphore smem_semaphore; 
        kittens::warp::init_semaphore(smem_semaphore, 0, 1);
        // *************************************************************************************************
        // Doing it this way would also work, but I want to illustrate the use of the cluster::expect, too.
        // kittens::tma::expect(smem_semaphore, shared_tile);
        // *************************************************************************************************
        kittens::everyone::tma::cluster::sync(); // ensure everyone has initialized their semaphore

        if(rank == 0 && threadIdx.x == 0) { // only one block issues the multicast load for everyone
            for(int j = 0; j < 4; j++) { // expect on the whole block
                if(threadIdx.x == 0) kittens::tma::cluster::expect(smem_semaphore, j, shared_tile);
            }
            if(threadIdx.x == 0) kittens::tma::cluster::load_async(shared_tile, input, {0, 0, 0, 0}, smem_semaphore, 0b1111);
        }

        kittens::wait(smem_semaphore, 0);
        kittens::warp::store(output, shared_tile, {0, 0, rank, 0});
        kittens::everyone::tma::cluster::sync();
    }
};

template<typename Ker, typename T, int H, int W, int NW, kittens::ducks::gl::all GL, typename... args>
static __global__ __cluster_dims__(4, 1, 1) void tmamulti_global_wrapper_2d(const __grid_constant__ GL input, const __grid_constant__ GL output) {
    Ker::template device_func<H, W, NW, GL, args...>(input, output);
}
template<typename test, int H, int W, int NUM_WORKERS, typename... args>
struct tmamulti_wrapper_2d {
    using dtype = gmem_dtype<test>; // defaults to bf16 in global memory if the test doesn't specify.
    static void run(test_data& results) {
        test_info this_result;
        this_result.label = generate_test_name<H, W, NUM_WORKERS, args...>(test::test_identifier);
        if constexpr (test::template valid<H, W, NUM_WORKERS, args...>::value) {
            constexpr int SIZE = H*W*256 * 4; // 4 for additional TMA dimension
            // initialize
            dtype *d_i, *d_o;
            std::vector<float> i_ref(SIZE);
            std::vector<float> o_ref(SIZE);
            initialize(&d_i, &d_o, i_ref, o_ref);
            // make descriptors
            using GL = typename kittens::gl<dtype, 1, 1, 64*H, 16*W, kittens::st<dtype, 16*H, 16*W>>;
            GL input(d_i, nullptr, nullptr, nullptr, nullptr);
            GL output(d_o, nullptr, nullptr, nullptr, nullptr);
            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                tmamulti_global_wrapper_2d<test), dtype, H, W, NUM_WORKERS, GL, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );
            tmamulti_global_wrapper_2d<test, dtype, H, W, NUM_WORKERS, GL, args...><<<4, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(input, output);
            // fill in correct results on cpu
            test::template host_func<H, W, NUM_WORKERS, GL, args...>(i_ref, o_ref);
            // check and cleanup
            this_result.result = validate(d_i, d_o, i_ref, o_ref, this_result.label, W*kittens::TILE_COL_DIM<dtype>);
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};
template<typename test, int MAX_H=8, int MAX_W=8, int NUM_WORKERS=1, typename... args>
using tmamulti_sweep_size_2d = loop_h<tmamulti_wrapper_2d, test, MAX_H, MAX_W, NUM_WORKERS, MAX_H, args...>;
template<template<typename> typename test, int MAX_H=8, int MAX_W=8, int NUM_WORKERS=1, typename... args>
struct tmamulti_sweep_gmem_type_2d {
    static void run(test_data &results) {
        tmamulti_sweep_size_2d<test<float>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
        tmamulti_sweep_size_2d<test<kittens::bf16>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
        tmamulti_sweep_size_2d<test<kittens::half>, MAX_H, MAX_W, NUM_WORKERS, args...>::run(results);
    }
};
template<template<typename> typename test, int MAX_H=8, int MAX_W=8, typename... args> using tmamulti_sweep_gmem_type_2d_warp = tmamulti_sweep_gmem_type_2d<test, MAX_H, MAX_W, 1, args...>;

void thread::memory::tile::tma_multicast::tests(test_data &results) {
    std::cout << " ----- Starting ops/thread/memory/tile/tma_multicast tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    tmamulti_sweep_gmem_type_2d_warp<test_load_multicast, SIZE, SIZE>::run(results);
    std::cout << std::endl;
}

#endif