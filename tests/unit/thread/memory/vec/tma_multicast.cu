#include "hip/hip_runtime.h"
#include "tma_multicast.cuh"

#ifdef TEST_THREAD_MEMORY_VEC_TMA_MULTICAST

template<typename T>
struct test_load_multicast { // load with TMA, write out normally
    using dtype = T;
    template<int S, int NW> using valid = std::bool_constant<NW == 1>;
    static inline const std::string test_identifier = std::is_same_v<T, kittens::bf16> ? "tma_multicast_load_vec_gmem=bf16" :
                                                      std::is_same_v<T, kittens::half> ? "tma_multicast_load_vec_gmem=half" :
                                                                                         "tma_multicast_load_vec_gmem=float";
    template<int S, int NW, kittens::ducks::gl::all GL> __host__ static void host_func(const std::vector<float> &i_ref, std::vector<float> &o_ref) {
        int SIZE_DIV_4 = i_ref.size()/4;
        for(int i = 0; i < SIZE_DIV_4; i++) {
            for(int j = 0; j < 4; j++) {
                o_ref[i+j*SIZE_DIV_4] = i_ref[i];
            }
        }
    }
    template<int S, int NW, kittens::ducks::gl::all GL>
    __device__ static void device_func(const GL &input, const GL &output) {
        extern __shared__ kittens::alignment_dummy __shm[]; // this is the CUDA shared memory
        kittens::tma_swizzle_allocator al((int*)&__shm[0]); 
        kittens::row_vec<kittens::st<dtype, 16*S, 16*S>> (&shared_vec) = al.allocate<kittens::row_vec<kittens::st<dtype, 16*S, 16*S>>>();
        int rank = blockIdx.x % 4;
        
        __shared__ kittens::semaphore smem_semaphore; 
        kittens::warp::init_semaphore(smem_semaphore, 0, 1);
        // *************************************************************************************************
        // Doing it this way would also work, but I want to illustrate the use of the cluster::expect, too.
        // if(threadIdx.x == 0) kittens::tma::expect<typeof(shared_vec)>(smem_semaphore);
        // *************************************************************************************************
        kittens::everyone::tma::cluster::sync(); // ensure everyone has initialized their semaphore

        if(rank == 0 && threadIdx.x == 0) { // only one block issues the multicast load for everyone
            for(int j = 0; j < 4; j++) { // expect on the whole block
                if(threadIdx.x == 0) kittens::tma::cluster::expect(smem_semaphore, j, shared_vec);
            }
            if(threadIdx.x == 0) kittens::tma::cluster::load_async(shared_vec, input, {0, 0, 0, 0}, smem_semaphore, 0b1111);
        }

        kittens::wait(smem_semaphore, 0);
        kittens::warp::store(output, shared_vec, {0, 0, rank, 0});
        kittens::everyone::tma::cluster::sync();
    }
};

template<typename Ker, typename T, int S, int NW, kittens::ducks::gl::all GL, typename... args>
static __global__ __cluster_dims__(4, 1, 1) void tmamulti_global_wrapper_1d(const __grid_constant__ GL input, const __grid_constant__ GL output) {
    Ker::template device_func<S, NW, GL, args...>(input, output);
}
template<typename test, int S, int NUM_WORKERS, typename... args>
struct tmamulti_wrapper_1d {
    using dtype = gmem_dtype<test>; // defaults to bf16 in global memory if the test doesn't specify.
    static void run(test_data& results) {
        test_info this_result;
        this_result.label = generate_test_name<S,NUM_WORKERS, args...>(test::test_identifier);
        if constexpr (test::template valid<S, NUM_WORKERS, args...>::value) {
            constexpr int SIZE = S*16 * 4; // 4 for additional TMA dimension
            // initialize
            dtype *d_i, *d_o;
            std::vector<float> i_ref(SIZE);
            std::vector<float> o_ref(SIZE);
            initialize(&d_i, &d_o, i_ref, o_ref);
            // make descriptors
            using GL = typename kittens::gl<dtype, 1, 1, 4, S*16, kittens::sv<dtype, 16*S>>;
            GL input(d_i, nullptr, nullptr, nullptr, nullptr);
            GL output(d_o, nullptr, nullptr, nullptr, nullptr);
            // run kernel
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                tmamulti_global_wrapper_1d<test), dtype, S, NUM_WORKERS, GL, args...>,
                hipFuncAttributeMaxDynamicSharedMemorySize,
                kittens::MAX_SHARED_MEMORY
            );
            tmamulti_global_wrapper_1d<test, dtype, S, NUM_WORKERS, GL, args...><<<4, NUM_WORKERS*32, kittens::MAX_SHARED_MEMORY>>>(input, output);
            // fill in correct results on cpu
            test::template host_func<S, NUM_WORKERS, GL, args...>(i_ref, o_ref);
            // check and cleanup
            this_result.result = validate(d_i, d_o, i_ref, o_ref, this_result.label, S*16);
        }
        else {
            this_result.result = test_result::INVALID;
        }
        results.push_back(this_result);
    }
};
template<typename test, int MAX_S, int NW, typename... args>
using tmamulti_sweep_size_1d = loop_s<tmamulti_wrapper_1d, test, MAX_S, NW, MAX_S, args...>;

template<template<typename> typename test, int MAX_S=8, int NUM_WORKERS=1, typename... args>
struct tmamulti_sweep_gmem_type_1d {
    static void run(test_data &results) {
        tmamulti_sweep_size_1d<test<float>, MAX_S, NUM_WORKERS, args...>::run(results);
        tmamulti_sweep_size_1d<test<kittens::bf16>, MAX_S, NUM_WORKERS, args...>::run(results);
        tmamulti_sweep_size_1d<test<kittens::half>, MAX_S, NUM_WORKERS, args...>::run(results);
    }
};
template<template<typename> typename test, int MAX_S=8, typename... args> using tmamulti_sweep_gmem_type_1d_warp = tmamulti_sweep_gmem_type_1d<test, MAX_S, 1, args...>;

void thread::memory::vec::tma_multicast::tests(test_data &results) {
    std::cout << " ----- Starting ops/thread/memory/vec/tma_multicast tests! -----\n" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;

    tmamulti_sweep_gmem_type_1d_warp<test_load_multicast, SIZE>::run(results);
    std::cout << std::endl;
}

#endif